#include "hip/hip_runtime.h"
/* -*- c++ -*- */

// CUB for reductions.
#include <hipcub/hipcub.hpp>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_util.h>
#include <gkyl_array_integrate.h>
#include <gkyl_array_integrate_priv.h>
}

__global__ static void
gkyl_array_integrate_set_ker_cu(struct gkyl_array_integrate *up, enum gkyl_array_integrate_op op, struct gkyl_basis basis)
{
  int ndim = basis.ndim, poly_order = basis.poly_order;

  if (op == GKYL_ARRAY_INTEGRATE_OP_NONE) {
    up->kernel = gkyl_array_integrate_none_ker_list_ser[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_ABS) {
    up->kernel = gkyl_array_integrate_abs_ker_list_ser[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_SQ) {
    up->kernel = gkyl_array_integrate_sq_ker_list_ser[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_SQ_WEIGHTED) {
    if (basis.b_type == GKYL_BASIS_MODAL_SERENDIPITY)
      up->kernel = gkyl_array_integrate_sq_weighted_ker_list_ser[ndim-1].kernels[poly_order-1];
    else if (basis.b_type == GKYL_BASIS_MODAL_GKHYBRID)
      up->kernel = gkyl_array_integrate_sq_weighted_ker_list_gkhyb[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_GRAD_SQ) {
    up->kernel = gkyl_array_integrate_gradsq_ker_list[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_GRADPERP_SQ) {
    up->kernel = gkyl_array_integrate_gradperpsq_ker_list[ndim-1].kernels[poly_order-1];
  }
  else if (op == GKYL_ARRAY_INTEGRATE_OP_EPS_GRADPERP_SQ) {
    up->kernel = gkyl_array_integrate_epsgradperpsq_ker_list[ndim-1].kernels[poly_order-1];
  } else {
    assert(false);
  }
}

struct gkyl_array_integrate*
gkyl_array_integrate_cu_dev_new(const struct gkyl_rect_grid *grid, const struct gkyl_basis *basis,
  int num_comp, enum gkyl_array_integrate_op op)
{
  // Allocate space for new updater.
  struct gkyl_array_integrate *up = (struct gkyl_array_integrate*) gkyl_malloc(sizeof(struct gkyl_array_integrate));

  up->op = op;
  up->num_basis = basis->num_basis;
  up->num_comp = num_comp;
  up->use_gpu = true;
  for (int d=0; d<grid->ndim; ++d) up->dxSq[d] = grid->dx[d]*grid->dx[d];

  assert(basis->poly_order > 0); // Need to check normalization for p=0.

  int ndim = basis->ndim;
  up->vol = 1.0;
  for (unsigned d=0; d<ndim; ++d)
    up->vol *= grid->dx[d]/2.0;

  // Copy struct to device.
  struct gkyl_array_integrate *up_cu = (struct gkyl_array_integrate*) gkyl_cu_malloc(sizeof(struct gkyl_array_integrate));
  gkyl_cu_memcpy(up_cu, up, sizeof(struct gkyl_array_integrate), GKYL_CU_MEMCPY_H2D);

  // Set the kernel.
  gkyl_array_integrate_set_ker_cu<<<1,1>>>(up_cu, op, *basis);

  up->on_dev = up_cu;

  return up;
}

template <unsigned int BLOCKSIZE>
__global__ void
array_integrate_blockRedAtomic_cub(struct gkyl_array_integrate *up, const struct gkyl_array *inp,
  double factor, const struct gkyl_array *weight, const struct gkyl_range range, struct gkyl_range weight_range, double *out)
{
  unsigned long linc = blockIdx.x*blockDim.x + threadIdx.x;

  // Specialize BlockReduce for type double.
  typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

  // Allocate temporary storage in shared memory.
  __shared__ typename BlockReduceT::TempStorage temp;

  int idx[GKYL_MAX_DIM];
  gkyl_sub_range_inv_idx(&range, linc, idx);
  long start = gkyl_range_idx(&range, idx);

  double outLocal[10]; // Set to max of 10 (e.g. heat flux tensor).
  for (unsigned int k=0; k<up->num_comp; ++k)
    outLocal[k] = 0.0;

  // Integrate in this cell
  if (linc < range.volume) {
    const double *fptr = (const double*) gkyl_array_cfetch(inp, start);

    const double *wptr = 0;
    if (weight) {
      int widx[GKYL_MAX_DIM];
      for (int d=0; d<weight_range.ndim; d++) widx[d] = idx[d]; 
      long linidx_w = gkyl_range_idx(&weight_range, widx);
      wptr = (const double*) gkyl_array_cfetch(weight, linidx_w);
    }

    up->kernel(up->dxSq, up->vol*factor, up->num_comp, up->num_basis, wptr, fptr, outLocal);
  }

  for (size_t k = 0; k < up->num_comp; ++k) {
    double bResult = 0;
    bResult = BlockReduceT(temp).Reduce(outLocal[k], hipcub::Sum());
    if (threadIdx.x == 0)
      atomicAdd(&out[k], bResult);
  }
}

void gkyl_array_integrate_advance_cu(gkyl_array_integrate *up, const struct gkyl_array *fin,
  double factor, const struct gkyl_array *weight, const struct gkyl_range *range, const struct gkyl_range *weight_range, double *out)
{
  gkyl_cu_memset(out, 0, up->num_comp*sizeof(double));

  const int nthreads = GKYL_DEFAULT_NUM_THREADS;
  int nblocks = gkyl_int_div_up(range->volume, nthreads);
  struct gkyl_array *weight_on_dev = NULL;
  struct gkyl_range weight_range_copy;
  if (weight) {
    weight_on_dev = weight->on_dev;
    weight_range_copy = *weight_range;
  }
  array_integrate_blockRedAtomic_cub<nthreads><<<nblocks, nthreads>>>(up->on_dev, fin->on_dev, factor, 
    weight_on_dev, *range, weight_range_copy, out);
  // device synchronize required because out may be host pinned memory
  hipDeviceSynchronize();
}
