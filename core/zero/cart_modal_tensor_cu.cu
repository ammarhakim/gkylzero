#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <assert.h>
#include <string.h>    
    
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_cart_modal_tensor_priv.h>
#include <gkyl_util.h>
}

__global__ void static
gkyl_cart_modal_tensor_cu_dev_kern(struct gkyl_basis *basis, int ndim, int poly_order)
{
  assert(ev_list[ndim].ev[poly_order]);

  basis->ndim = ndim;
  basis->poly_order = poly_order;
  basis->num_basis = pow(poly_order+1, ndim);
  basis->b_type = GKYL_BASIS_MODAL_TENSOR;  
  
  // function pointers
  basis->eval = ev_list[ndim].ev[poly_order];
  basis->eval_expand = eve_list[ndim].ev[poly_order];
  basis->eval_grad_expand = eveg_list[ndim].ev[poly_order];
  basis->flip_odd_sign = fos_list[ndim].fs[poly_order];
  basis->flip_even_sign = fes_list[ndim].fs[poly_order];
  basis->node_list = nl_list[ndim].nl[poly_order];
  basis->nodal_to_modal = n2m_list[ndim].n2m[poly_order];
  basis->quad_nodal_to_modal = qn2m_list[ndim].n2m[poly_order];
  basis->modal_to_quad_nodal = m2qn_list[ndim].n2m[poly_order];
}

void
gkyl_cart_modal_tensor_cu_dev(struct gkyl_basis *basis, int ndim, int poly_order)
{
  assert(ndim>0 && ndim<=6);

  struct gkyl_basis ho_basis;

  strcpy(ho_basis.id, "tensor");
  // this copy needs to be done here as the strcpy needed in the
  // "type" field can't be done on the device
  gkyl_cu_memcpy(basis, &ho_basis, sizeof(struct gkyl_basis),
    GKYL_CU_MEMCPY_H2D);
  
  gkyl_cart_modal_tensor_cu_dev_kern<<<1,1>>>(basis, ndim, poly_order);
}

struct gkyl_basis *
gkyl_cart_modal_tensor_cu_dev_new(int ndim, int poly_order)
{
  struct gkyl_basis *basis = (struct gkyl_basis *) gkyl_cu_malloc(sizeof(struct gkyl_basis));
  gkyl_cart_modal_tensor_cu_dev(basis, ndim, poly_order);
  return basis;
}
