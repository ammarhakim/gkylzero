
#include <hip/hip_runtime.h>
#ifdef GKYL_HAVE_CUDSS

#include <cuDSS.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_mat_triples.h>
#include <gkyl_array_ops.h>
#include <gkyl_range.h>
#include <gkyl_rect_grid.h>
#include <gkyl_util.h>
#include <gkyl_culinsolver_ops.h>
}

#define checkCUDSS(call, status, msg) \
do { \
    status = call; \
    if (status != CUDSS_STATUS_SUCCESS) { \
        fprintf(stderr, "Example FAILED: CUDSS call ended unsuccessfully with status = %d, details: " #msg "\n", status); \
        exit(EXIT_FAILURE); \
    } \
} while(0);

struct gkyl_culinsolver_prob {
  double *rhs_ho, *rhs_cu; // right-hand side vector.
  double *x_ho, *x_cu; // solution vector.
  int nprob; // number of problems to solve.
  int mrow, ncol; // A is a mrow x ncol matrix.
  int nnz; // number of non-zero entries in A.
  int nrhs; // number of columns in B (B is an mrow x nrhs matrix).

  cudaStream_t stream; // CUDA stream cuDSS runs on.
  cudssHandle_t handle; // cuDSS handle.
  cudssConfig_t *solverConfig;
  cudssData_t *solverData;

  cudssMatrix_t *A; // cuDSS object holding the LHS matrix.
  cudssMatrix_t *x, *b; // cuDSS objects holding the unknowns vector and RHS vector.

  // Arrays used to populate the LHS matrix in CSR format.
  double *csr_val_cu;
  int *csr_rowptr_cu, *csr_colind_cu;
};

gkyl_culinsolver_prob*
gkyl_culinsolver_prob_new(int nprob, int mrow, int ncol, int nrhs)
{
  struct gkyl_culinsolver_prob *prob = (struct gkyl_culinsolver_prob*) gkyl_malloc(sizeof(*prob));

  prob->nprob = nprob;
  prob->mrow = mrow;
  prob->ncol = ncol;
  prob->nrhs = nrhs;

  prob->rhs_ho = (double*) gkyl_malloc(nprob * nrhs * mrow * sizeof(double));
  prob->rhs_cu = (double*) gkyl_cu_malloc(nprob * nrhs * mrow * sizeof(double));
  prob->x_ho = (double*) gkyl_malloc(nprob * nrhs * mrow * sizeof(double));
  prob->x_cu = (double*) gkyl_cu_malloc(nprob * nrhs * mrow * sizeof(double));

  cudssStatus_t status = CUDSS_STATUS_SUCCESS;

  /* Create a CUDA stream */
  prob->stream = NULL;
  checkCuda(cudaStreamCreate(&prob->stream));

  /* Creating the cuDSS library handle */
  checkCUDSS(cudssCreate(&prob->handle), status, "cudssCreate");

  /* (optional) Setting the custom stream for the library handle */
  checkCUDSS(cudssSetStream(prob->handle, prob->stream), status, "cudssSetStream");

  /* Creating cuDSS solver configuration and data objects */
  prob->solverConfig = (cudssConfig_t *) gkyl_malloc(nprob * sizeof(cudssConfig_t));
  prob->solverData = (cudssData_t *) gkyl_malloc(nprob * sizeof(cudssData_t));
  for (int i=0; i<nprob; i++) {
    checkCUDSS(cudssConfigCreate(&prob->solverConfig[i]), status, "cudssConfigCreate");
    checkCUDSS(cudssDataCreate(prob->handle, &prob->solverData[i]), status, "cudssDataCreate");
  }

  /* Create matrix objects for the right-hand side b and solution x (as dense matrices). */
  int64_t mrow_64 = mrow, ncol_64 = ncol;
  int ldb = ncol_64, ldx = mrow_64;
  for (int i=0; i<nprob * nrhs * mrow; i++)
    prob->rhs_ho[i] = 1.0;
  gkyl_cu_memcpy(prob->rhs_cu, prob->rhs_ho, nprob * nrhs * mrow * sizeof(double), GKYL_CU_MEMCPY_H2D);
  gkyl_cu_memcpy(prob->x_cu, prob->rhs_ho, nprob * nrhs * mrow * sizeof(double), GKYL_CU_MEMCPY_H2D);

  prob->b = (cudssMatrix_t *) gkyl_malloc(nprob * sizeof(cudssMatrix_t));
  prob->x = (cudssMatrix_t *) gkyl_malloc(nprob * sizeof(cudssMatrix_t));

  for (int i=0; i<nprob; i++) {
    long off = i * nrhs * mrow;
    checkCUDSS(cudssMatrixCreateDn(&prob->b[i], ncol_64, nrhs, ldb, prob->rhs_cu+off, CUDA_R_64F, CUDSS_LAYOUT_COL_MAJOR),
      status, "cudssMatrixCreateDn for b");
    checkCUDSS(cudssMatrixCreateDn(&prob->x[i], mrow_64, nrhs, ldx, prob->x_cu+off, CUDA_R_64F, CUDSS_LAYOUT_COL_MAJOR),
      status, "cudssMatrixCreateDn for x");
  }

  return prob;
}

void
gkyl_culinsolver_amat_from_triples(struct gkyl_culinsolver_prob *prob, struct gkyl_mat_triples **tri)
{
  prob->nnz = gkyl_mat_triples_size(tri[0]);
  for (size_t k=0; k<prob->nprob; k++) {
    assert(gkyl_mat_triples_size(tri[k]) == prob->nnz);  // No. of nonzeros must be the same for every problem.
    assert(gkyl_mat_triples_is_rowmaj(tri[k]));  // Triples must be in rowmaj order for cusolver.
  }

  // Convert triples to CSR arrays on device.
  // Use CSR format
  double *csr_val = (double*) gkyl_malloc(prob->nprob*prob->nnz*sizeof(double)); // non-zero matrix elements.
  int *csr_colind = (int*) gkyl_malloc(sizeof(int)*prob->nnz); // col index of entries in csrvalA.
  int *csr_rowptr = (int*) gkyl_malloc(sizeof(int)*(prob->mrow+1)); // 1st entry of each row as index in csrvalA.

  bool *csr_rowptr_assigned = (bool*) gkyl_malloc(sizeof(bool)*prob->mrow);
  for (size_t i=0; i<prob->mrow; i++) csr_rowptr_assigned[i] = false;

  // Sorted (row-major order) keys (linear indices to flattened matrix).
  for (size_t k=0; k<prob->nprob; k++) {
    gkyl_mat_triples_iter *iter = gkyl_mat_triples_iter_new(tri[k]);
    for (size_t i=0; i<prob->nnz; ++i) {
      gkyl_mat_triples_iter_next(iter); // bump iterator.
      struct gkyl_mtriple mt = gkyl_mat_triples_iter_at(iter);
      size_t idx[2] = { mt.row, mt.col };

      csr_val[k*prob->nnz+i] = mt.val;
      if (k==0) {
        csr_colind[i] = idx[1];
        if (!csr_rowptr_assigned[idx[0]]) {
          csr_rowptr[idx[0]] = i;
          csr_rowptr_assigned[idx[0]] = true;
        }
      }
    }
    gkyl_mat_triples_iter_release(iter);
  }
  csr_rowptr[prob->mrow] = prob->nnz;
  gkyl_free(csr_rowptr_assigned);

  // Copy arrays to device.
  prob->csr_val_cu = (double*) gkyl_cu_malloc(prob->nprob*prob->nnz*sizeof(double)); // Non-zero matrix elements.
  prob->csr_colind_cu = (int*) gkyl_cu_malloc(sizeof(int)*prob->nnz); // Col index of entries in csrvalA.
  prob->csr_rowptr_cu = (int*) gkyl_cu_malloc(sizeof(int)*(prob->mrow+1)); // 1st entry of each row as index in csrvalA.
  gkyl_cu_memcpy(prob->csr_val_cu, csr_val, prob->nprob*prob->nnz*sizeof(double), GKYL_CU_MEMCPY_H2D);
  gkyl_cu_memcpy(prob->csr_colind_cu, csr_colind, sizeof(int)*prob->nnz, GKYL_CU_MEMCPY_H2D);
  gkyl_cu_memcpy(prob->csr_rowptr_cu, csr_rowptr, sizeof(int)*(prob->mrow+1), GKYL_CU_MEMCPY_H2D);

  // Create a matrix object for the sparse input matrix.
  cudssStatus_t status = CUDSS_STATUS_SUCCESS;
//  cudssMatrixType_t mtype     = CUDSS_MTYPE_SPD;
  cudssMatrixType_t mtype     = CUDSS_MTYPE_GENERAL;
  cudssMatrixViewType_t mview = CUDSS_MVIEW_UPPER;
  cudssIndexBase_t base       = CUDSS_BASE_ZERO;
  prob->A = (cudssMatrix_t *) gkyl_malloc(prob->nprob * sizeof(cudssMatrix_t));
  for (int i=0; i<prob->nprob; i++) {
    long off = i * prob->nnz;

    checkCUDSS(cudssMatrixCreateCsr(&prob->A[i], prob->mrow, prob->ncol, prob->nnz, prob->csr_rowptr_cu, NULL,
      prob->csr_colind_cu, prob->csr_val_cu+off, CUDA_R_32I, CUDA_R_64F, mtype, mview,
      base), status, "cudssMatrixCreateCsr");
  
    // Symbolic factorization.
    checkCUDSS(cudssExecute(prob->handle, CUDSS_PHASE_ANALYSIS, prob->solverConfig[i], prob->solverData[i],
      prob->A[i], prob->x[i], prob->b[i]), status, "cudssExecute for analysis");
  
    // Factorization.
    checkCUDSS(cudssExecute(prob->handle, CUDSS_PHASE_FACTORIZATION, prob->solverConfig[i],
      prob->solverData[i], prob->A[i], prob->x[i], prob->b[i]), status, "cudssExecute for factor");
  }

  gkyl_free(csr_val);
  gkyl_free(csr_colind);
  gkyl_free(csr_rowptr);
}

void
gkyl_culinsolver_brhs_from_triples(struct gkyl_culinsolver_prob *prob, gkyl_mat_triples *tri)
{
  long nnz_rhs = gkyl_mat_triples_size(tri);  // Number of non-zero entries in RHS matrix B.

  // Sorted (column-major order) keys (linear indices to flattened matrix).
  gkyl_mat_triples_iter *iter = gkyl_mat_triples_iter_new(tri);
  for (size_t i=0; i<nnz_rhs; i++) {
    gkyl_mat_triples_iter_next(iter); // bump iterator
    struct gkyl_mtriple mt = gkyl_mat_triples_iter_at(iter);
    prob->rhs_ho[i] = mt.val;
  }
  gkyl_mat_triples_iter_release(iter);

  gkyl_cu_memcpy(prob->rhs_cu, prob->rhs_ho, prob->nprob*prob->mrow*prob->nrhs*sizeof(double), GKYL_CU_MEMCPY_H2D);

  cudssStatus_t status = CUDSS_STATUS_SUCCESS;
  for (size_t i=0; i<prob->nprob; i++) {
    long off = i * prob->mrow*prob->nrhs;
    checkCUDSS(cudssMatrixSetValues(prob->b[i], prob->rhs_cu+off),
      status, "cudssMatrixSetValues for setting brhs_from_triples");
  }
}

void
gkyl_culinsolver_solve(struct gkyl_culinsolver_prob *prob)
{
  cudssStatus_t status = CUDSS_STATUS_SUCCESS;

  for (size_t i=0; i<prob->nprob; i++) {
    checkCUDSS(cudssExecute(prob->handle, CUDSS_PHASE_SOLVE, prob->solverConfig[i], prob->solverData[i],
      prob->A[i], prob->x[i], prob->b[i]), status, "cudssExecute for solve");
  }
}

void
gkyl_culinsolver_sync(struct gkyl_culinsolver_prob *prob)
{
  cudaStreamSynchronize(prob->stream);
}

void
gkyl_culinsolver_finish_host(struct gkyl_culinsolver_prob *prob)
{
  //cudaStreamSynchronize(prob->stream); // not needed when using blocking stream
  gkyl_cu_memcpy(prob->x_ho, prob->x_cu, prob->nprob*prob->mrow*prob->nrhs*sizeof(double), GKYL_CU_MEMCPY_D2H);
}

void
gkyl_culinsolver_clear_rhs(struct gkyl_culinsolver_prob *prob, double val)
{
  gkyl_cu_memset(prob->rhs_cu, val, prob->nprob*prob->mrow*prob->nrhs*sizeof(double));
}

double*
gkyl_culinsolver_get_rhs_ptr(struct gkyl_culinsolver_prob *prob, long loc)
{
  return prob->rhs_cu+loc;
}

double*
gkyl_culinsolver_get_sol_ptr(struct gkyl_culinsolver_prob *prob, long loc)
{
  return prob->x_cu+loc;
}

double
gkyl_culinsolver_get_sol_lin(struct gkyl_culinsolver_prob *prob, long loc)
{
  return prob->x_ho[loc];
}

void
gkyl_culinsolver_prob_release(struct gkyl_culinsolver_prob *prob)
{
  cudssStatus_t status = CUDSS_STATUS_SUCCESS;

  for (size_t i=0; i<prob->nprob; i++) {
    checkCUDSS(cudssMatrixDestroy(prob->A[i]), status, "cudssMatrixDestroy for A");
    checkCUDSS(cudssMatrixDestroy(prob->b[i]), status, "cudssMatrixDestroy for b");
    checkCUDSS(cudssMatrixDestroy(prob->x[i]), status, "cudssMatrixDestroy for x");
    checkCUDSS(cudssDataDestroy(prob->handle, prob->solverData[i]), status, "cudssDataDestroy");
    checkCUDSS(cudssConfigDestroy(prob->solverConfig[i]), status, "cudssConfigDestroy");
  }
  checkCUDSS(cudssDestroy(prob->handle), status, "cudssHandleDestroy");
  gkyl_free(prob->A);
  gkyl_free(prob->b);
  gkyl_free(prob->x);
  gkyl_free(prob->solverData);
  gkyl_free(prob->solverConfig);

  gkyl_cu_free(prob->csr_colind_cu);
  gkyl_cu_free(prob->csr_rowptr_cu);
  gkyl_cu_free(prob->csr_val_cu);

  checkCuda(cudaStreamSynchronize(prob->stream));
  cudaStreamDestroy(prob->stream);

  gkyl_free(prob->rhs_ho);
  gkyl_free(prob->x_ho);
  gkyl_cu_free(prob->rhs_cu);
  gkyl_cu_free(prob->x_cu);

  gkyl_free(prob);
}

// End ifdef GKYL_HAVE_CUDSS statement.
#endif
