#include "hip/hip_runtime.h"
/* -*- c -*- */

#include <stdio.h>

extern "C" {
#include <gkyl_array_ops.h>
#include <gkyl_util.h>
#include <gkyl_alloc.h>
  int cu_array_test_and_flip_sign( struct gkyl_array *arr);
  void set_array_copy_fn(struct gkyl_array_copy_func *fn);
}

GKYL_CU_DH static void
buffer_fn_cu(size_t nc, double *out, const double *inp, void *ctx)
{
  for (size_t i=0; i<nc; ++i)
    out[i] = 2*inp[i];
}

__global__
void ker_cu_array_test_and_flip_sign( struct gkyl_array *arr, int *nfail)
{
  *nfail = 0;
  
  GKYL_CU_CHECK( arr->type == GKYL_DOUBLE, nfail );
  GKYL_CU_CHECK( arr->elemsz ==sizeof(double), nfail );
  GKYL_CU_CHECK( arr->ncomp == 1, nfail );
  GKYL_CU_CHECK( arr->size == 20, nfail );
  
  double *data = (double *) arr->data;
  for (unsigned i=0; i<arr->size; ++i) {
    GKYL_CU_CHECK( data[i] == (i+0.5)*0.1, nfail );
    data[i] *= -1;
  }
}

__global__ void
ker_set_array_copy_fn(struct gkyl_array_copy_func *fn)
{
  fn->func = buffer_fn_cu;
  fn->ctx = 0;
}

int
cu_array_test_and_flip_sign( struct gkyl_array *arr)
{
  int *nfail_dev = (int *) gkyl_cu_malloc(sizeof(int));
  ker_cu_array_test_and_flip_sign<<<1,1>>>(arr, nfail_dev);

  int nfail;
  gkyl_cu_memcpy(&nfail, nfail_dev, sizeof(int), GKYL_CU_MEMCPY_D2H);
  gkyl_cu_free(nfail_dev);

  return nfail;
}

void
set_array_copy_fn(struct gkyl_array_copy_func *fn)
{
  ker_set_array_copy_fn<<<1,1>>>(fn);  
}
