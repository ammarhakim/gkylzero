#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_gyrokinetic_pol_density.h>
#include <gkyl_gyrokinetic_pol_density_priv.h>
}

// CUDA kernel to set device pointers to kernels.
__global__ static void
gkyl_gk_pol_den_set_cu_ker_ptrs(struct gkyl_gyrokinetic_pol_density_kernels *kernels,
  struct gkyl_basis cbasis)
{
  int pdim = cbasis.ndim;
  enum gkyl_basis_type b_type = cbasis.b_type;
  int poly_order = cbasis.poly_order;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      kernels->pol_den = gk_pol_density_kern_list_ser[pdim-1].kernels[poly_order-1];
      break;
    default:
      assert(false);
  }
};

void
gk_pol_den_choose_kernel_cu(struct gkyl_gyrokinetic_pol_density_kernels *kernels,
  struct gkyl_basis cbasis)
{
  gkyl_gk_pol_den_set_cu_ker_ptrs<<<1,1>>>(kernels, cbasis);
}

__global__ static void
gkyl_gyrokinetic_pol_density_advance_cu_ker(
  struct gkyl_gyrokinetic_pol_density_kernels *kers,
  const struct gkyl_rect_grid grid, const struct gkyl_range conf_range,
  const struct gkyl_array* GKYL_RESTRICT pol_weight, const struct gkyl_array* GKYL_RESTRICT phi,
  struct gkyl_array* GKYL_RESTRICT npol)
{
  int cidx[GKYL_MAX_CDIM];

  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < conf_range.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&conf_range, tid, cidx);

    long linidx = gkyl_range_idx(&conf_range, cidx);
    const double *pol_weight_d = (const double*) gkyl_array_cfetch(pol_weight, linidx);
    const double *phi_d = (const double*) gkyl_array_cfetch(phi, linidx);
    double *npol_d = (double*) gkyl_array_fetch(npol, linidx);

    // Compute the polarization density.
    kers->pol_den(grid.dx, pol_weight_d, phi_d, npol_d);
  }
}

void
gkyl_gyrokinetic_pol_density_advance_cu(gkyl_gyrokinetic_pol_density* up,
  const struct gkyl_range *conf_rng, const struct gkyl_array *GKYL_RESTRICT pol_weight,
  const struct gkyl_array *GKYL_RESTRICT phi, struct gkyl_array *GKYL_RESTRICT npol)
{
  int nblocks = conf_rng->nblocks, nthreads = conf_rng->nthreads;

  gkyl_gyrokinetic_pol_density_advance_cu_ker<<<nblocks, nthreads>>>
    (up->kernels, up->grid, *conf_rng, pol_weight->on_dev, phi->on_dev, npol->on_dev);
}
