#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_dg_calc_gk_rad_vars.h>
#include <gkyl_dg_calc_gk_rad_vars_priv.h>
#include <gkyl_util.h>
}

__global__ void
gkyl_dg_calc_gk_rad_vars_nu_advance_cu_kernel(struct gkyl_dg_calc_gk_rad_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  double a, double alpha, double beta, double gamma, double v0, 
  struct gkyl_array* vnu_surf, struct gkyl_array* vnu, 
  struct gkyl_array* vsqnu_surf, struct gkyl_array* vsqnu)
{
  int pdim = up->pdim;
  int cdim = up->cdim;
  int idx[GKYL_MAX_DIM], idx_vel[2];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);

    for (int d=cdim; d<pdim; d++) idx_vel[d-cdim] = idx[d];

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_vel = gkyl_range_idx(&up->vel_map->local_vel, idx_vel);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double *bmag_d = (const double*) gkyl_array_cfetch(up->gk_geom->bmag, loc_conf);

    double* vnu_surf_d = (double*) gkyl_array_fetch(vnu_surf, loc_phase);
    double* vnu_d = (double*) gkyl_array_fetch(vnu, loc_phase);
    double* vsqnu_surf_d = (double*) gkyl_array_fetch(vsqnu_surf, loc_phase);  
    double* vsqnu_d = (double*) gkyl_array_fetch(vsqnu, loc_phase);   
    const double *vmap_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap, loc_vel);
    const double *vmapSq_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap_sq, loc_vel);

    up->rad_nu_vpar(vmap_d, vmapSq_d, up->charge, up->mass, 
      a, alpha, beta, gamma, v0, bmag_d, vnu_surf_d, vnu_d);
    up->rad_nu_mu(vmap_d, vmapSq_d, up->charge, up->mass, 
      a, alpha, beta, gamma, v0, bmag_d, vsqnu_surf_d, vsqnu_d);
  }  
}

// Host-side wrapper for radiation drag coefficient calculation
void 
gkyl_dg_calc_gk_rad_vars_nu_advance_cu(const struct gkyl_dg_calc_gk_rad_vars *up,
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range,
  double a, double alpha, double beta, double gamma, double v0, 
  struct gkyl_array* vnu_surf, struct gkyl_array* vnu, 
  struct gkyl_array* vsqnu_surf, struct gkyl_array* vsqnu)
{
  int nblocks = phase_range->nblocks;
  int nthreads = phase_range->nthreads;
  gkyl_dg_calc_gk_rad_vars_nu_advance_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *phase_range, a, alpha, beta, gamma, v0,
    vnu_surf->on_dev, vnu->on_dev, vsqnu_surf->on_dev, vsqnu->on_dev);
}

__global__ void
gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu_kernel(struct gkyl_dg_calc_gk_rad_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  const struct gkyl_gk_rad_drag* vnu_surf, const struct gkyl_gk_rad_drag* vnu, 
  const struct gkyl_gk_rad_drag* vsqnu_surf, const struct gkyl_gk_rad_drag* vsqnu,
  const struct gkyl_array* n_elc_rad, const struct gkyl_array* n_elc,
  const struct gkyl_array *nI, 
  struct gkyl_array* nvnu_surf, struct gkyl_array* nvnu, 
  struct gkyl_array* nvsqnu_surf, struct gkyl_array* nvsqnu,
  struct gkyl_array* vtsq_min_normalized, struct gkyl_array* vtsq)
{
  int cdim = up->cdim;
  int idx[GKYL_MAX_DIM];
  
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_phase = gkyl_range_idx(&phase_range, idx);
    const double* vtsq_d = (const double*) gkyl_array_cfetch(vtsq, loc_conf);

    const double* ne = (const double*)gkyl_array_cfetch(n_elc, loc_conf);
    double ne_cell_avg = ne[0]/pow(2.0, cdim/2.0);
    
    // Find nearest index
    int ne_idx = gkyl_dg_rad_gyrokinetic_find_nearest_idx(n_elc_rad, ne_cell_avg);

    const double* vtsq_min_d = (const double*) gkyl_array_cfetch(vtsq_min_normalized, ne_idx);
    if ( vtsq_d[0] > vtsq_min_d[0] ) {            
      const double* vnu_surf_d = (const double*) gkyl_array_cfetch(vnu_surf[ne_idx].arr, loc_phase);
      const double* vnu_d = (const double*) gkyl_array_cfetch(vnu[ne_idx].arr, loc_phase);
      const double* vsqnu_surf_d = (const double*) gkyl_array_cfetch(vsqnu_surf[ne_idx].arr, loc_phase);  
      const double* vsqnu_d = (const double*) gkyl_array_cfetch(vsqnu[ne_idx].arr, loc_phase);   
      
      const double *nI_d = (const double*) gkyl_array_cfetch(nI, loc_conf);
      
      double* nvnu_surf_d = (double*) gkyl_array_fetch(nvnu_surf, loc_phase);
      double* nvnu_d = (double*) gkyl_array_fetch(nvnu, loc_phase);
      double* nvsqnu_surf_d = (double*) gkyl_array_fetch(nvsqnu_surf, loc_phase);  
      double* nvsqnu_d = (double*) gkyl_array_fetch(nvsqnu, loc_phase);   
      
      up->rad_nI_nu(vnu_surf_d, vnu_d, vsqnu_surf_d, vsqnu_d, nI_d, 
		    nvnu_surf_d, nvnu_d, nvsqnu_surf_d, nvsqnu_d);
    }
  }  
}

// Host-side wrapper for sum_s n_{i_s} nu_s(v) calculation for a given input n_{i_s} and nu_s(v)
void 
gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu(const struct gkyl_dg_calc_gk_rad_vars *up,
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range, 
  const struct gkyl_gk_rad_drag* vnu_surf, const struct gkyl_gk_rad_drag* vnu, 
  const struct gkyl_gk_rad_drag* vsqnu_surf, const struct gkyl_gk_rad_drag* vsqnu,
  const struct gkyl_array* n_elc_rad, const struct gkyl_array* n_elc,
  const struct gkyl_array *nI, 
  struct gkyl_array* nvnu_surf, struct gkyl_array* nvnu, 
  struct gkyl_array* nvsqnu_surf, struct gkyl_array* nvsqnu,
  struct gkyl_array* vtsq_min_normalized, struct gkyl_array* vtsq)
{
  int nblocks = phase_range->nblocks;
  int nthreads = phase_range->nthreads;
  gkyl_dg_calc_gk_rad_vars_nI_nu_advance_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *phase_range,
    vnu_surf->on_dev, vnu->on_dev, vsqnu_surf->on_dev, vsqnu->on_dev, 
    n_elc_rad->on_dev, n_elc->on_dev, nI->on_dev, 
    nvnu_surf->on_dev, nvnu->on_dev, nvsqnu_surf->on_dev, nvsqnu->on_dev,
    vtsq_min_normalized->on_dev, vtsq->on_dev);
}

// CUDA kernel to set device pointers to gyrokinetic radiation vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_gk_rad_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_gk_rad_vars *up, 
  int cdim, int vdim, int poly_order)
{
  up->rad_nu_vpar = choose_rad_gyrokinetic_nu_vpar_kern(cdim, vdim, poly_order);
  up->rad_nu_mu = choose_rad_gyrokinetic_nu_mu_kern(cdim, vdim, poly_order);
  up->rad_nI_nu = choose_rad_gyrokinetic_nI_nu_kern(cdim, vdim, poly_order);
}

gkyl_dg_calc_gk_rad_vars*
gkyl_dg_calc_gk_rad_vars_cu_dev_new(const struct gkyl_rect_grid *phase_grid, 
  const struct gkyl_basis *conf_basis, const struct gkyl_basis *phase_basis, double charge,
  double mass, const struct gk_geometry *gk_geom, const struct gkyl_velocity_map *vel_map)
{
  struct gkyl_dg_calc_gk_rad_vars *up = (struct gkyl_dg_calc_gk_rad_vars*) gkyl_malloc(sizeof(*up));

  up->phase_grid = *phase_grid;
  int cdim = conf_basis->ndim;
  int pdim = phase_basis->ndim;
  int vdim = pdim - cdim;
  int poly_order = phase_basis->poly_order;
  up->cdim = cdim;
  up->pdim = pdim;

  up->charge = charge;
  up->mass = mass;

  // Acquire pointers to on_dev objects so memcpy below copies those too.
  struct gk_geometry *geom_ho = gkyl_gk_geometry_acquire(gk_geom);
  struct gkyl_velocity_map *vel_map_ho = gkyl_velocity_map_acquire(vel_map);
  up->gk_geom = geom_ho->on_dev;
  up->vel_map = vel_map_ho->on_dev;

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_gk_rad_vars *up_cu = (struct gkyl_dg_calc_gk_rad_vars*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_gk_rad_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_gk_rad_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, cdim, vdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;

  // Updater should store host pointers.
  up->gk_geom = geom_ho; 
  up->vel_map = vel_map_ho; 
  
  return up;
}
