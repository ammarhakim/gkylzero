#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array.h>
#include <gkyl_array_ops.h>
#include <gkyl_dg_bin_ops.h>
#include <gkyl_dg_cx.h>
#include <gkyl_dg_cx_priv.h>
#include <gkyl_util.h>
#include <gkyl_const.h>
}

// CUDA kernel to set device pointers to kernels.
__global__ static void
gkyl_dg_cx_set_cu_dev_ptrs(struct gkyl_dg_cx *up, struct gkyl_basis cbasis)
{
  up->react_rate = choose_kern(cbasis);
};

__global__ static void
gkyl_cx_react_rate_cu_ker(struct gkyl_dg_cx *up, const struct gkyl_range conf_rng, 
  const struct gkyl_array *maxwellian_moms_ion, const struct gkyl_array *maxwellian_moms_neut,
  const struct gkyl_array *upar_b_i,  double vt_sq_ion_min, double vt_sq_neut_min,
  struct gkyl_array *coef_cx, double a, double b)
{
  int cidx[GKYL_MAX_CDIM];
  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < conf_rng.volume; tid += blockDim.x*gridDim.x) {

    gkyl_sub_range_inv_idx(&conf_rng, tid, cidx);
    long linidx = gkyl_range_idx(&conf_rng, cidx);

    const double *maxwellian_moms_ion_d = (const double*) gkyl_array_cfetch(maxwellian_moms_ion, linidx);
    const double *maxwellian_moms_neut_d = (const double*) gkyl_array_cfetch(maxwellian_moms_neut, linidx);
    const double *upar_b_i_d = (const double*) gkyl_array_cfetch(upar_b_i, linidx);

    double *coef_cx_d = (double*) gkyl_array_fetch(coef_cx, linidx);

    // call the cx kernel
    double cflr = up->react_rate(a, b, vt_sq_ion_min, vt_sq_neut_min, 
      maxwellian_moms_ion_d, maxwellian_moms_neut_d, upar_b_i_d, coef_cx_d);
  }
}

void gkyl_dg_cx_coll_cu(const struct gkyl_dg_cx *up, 
  struct gkyl_array *maxwellian_moms_ion, struct gkyl_array *maxwellian_moms_neut,
  struct gkyl_array *upar_b_i, struct gkyl_array *coef_cx, struct gkyl_array *cflrate)
{  
  gkyl_cx_react_rate_cu_ker<<<up->conf_rng->nblocks, up->conf_rng->nthreads>>>(up->on_dev, *up->conf_rng,
    maxwellian_moms_ion->on_dev, maxwellian_moms_neut->on_dev, upar_b_i->on_dev, 
    up->vt_sq_ion_min, up->vt_sq_neut_min, coef_cx->on_dev, up->a, up->b);
}

gkyl_dg_cx*
gkyl_dg_cx_cu_dev_new(struct gkyl_dg_cx_inp *inp)
{
  gkyl_dg_cx *up = (struct gkyl_dg_cx*) gkyl_malloc(sizeof(*up));

  up->cbasis = inp->cbasis;
  up->conf_rng = inp->conf_rng;
  up->type_ion = inp->type_ion;
  up->vt_sq_ion_min = inp->vt_sq_ion_min;
  up->vt_sq_neut_min = inp->vt_sq_neut_min;

  fit_param(up->type_ion, &up->a, &up->b);
  
  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_cx *up_cu = (struct gkyl_dg_cx*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_cx), GKYL_CU_MEMCPY_H2D);

  gkyl_dg_cx_set_cu_dev_ptrs<<<1,1>>>(up_cu, *up->cbasis);

  // Set parent on_dev pointer.
  up->on_dev = up_cu;
  
  return up;
}
