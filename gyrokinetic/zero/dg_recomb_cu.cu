#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array.h>
#include <gkyl_array_ops.h>
#include <gkyl_dg_bin_ops.h>
#include <gkyl_dg_recomb.h>
#include <gkyl_dg_recomb_priv.h>
#include <gkyl_util.h>
#include <gkyl_const.h>
}

__global__ static void
gkyl_recomb_react_rate_cu_ker(const struct gkyl_dg_recomb *up, 
  const struct gkyl_range conf_rng, const struct gkyl_range adas_rng, const struct gkyl_basis *adas_basis, 
  const struct gkyl_array *prim_vars_elc, struct gkyl_array *coef_recomb, 
  struct gkyl_array *recomb_data, double mass_elc, double elem_charge, 
  double maxLogTe, double minLogTe, double dlogTe, int resTe,  
  double maxLogM0, double minLogM0, double dlogM0, int resM0)
{
  int cidx[GKYL_MAX_CDIM];
  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < conf_rng.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&conf_rng, tid, cidx);
    long loc = gkyl_range_idx(&conf_rng, cidx);
    long nc = coef_recomb->ncomp;

    int cdim = conf_rng.ndim;
    const double *prim_vars_elc_d = (const double*) gkyl_array_cfetch(prim_vars_elc, loc);
    double *coef_recomb_d = (double*) gkyl_array_fetch(coef_recomb, loc);

    //Find nearest neighbor for n, Te in ADAS interpolated data
    double cell_av_fac = pow(1.0/sqrt(2.0),cdim);
    double m0_elc_av = prim_vars_elc_d[0]*cell_av_fac;
    double temp_elc_av = prim_vars_elc_d[2*nc]*cell_av_fac*mass_elc/elem_charge;
    double log_Te_av = log10(temp_elc_av);
    double log_m0_av = log10(m0_elc_av);
    int m0_idx, t_idx;
    double cell_vals_2d[2];
    double cell_center;
    
    if (log_Te_av < minLogTe) {
      t_idx=1;
      log_Te_av = minLogTe;
    }
    else if (log_Te_av > maxLogTe) {
      t_idx=resTe;
      log_Te_av = maxLogTe;
    }
    else t_idx = (log_Te_av - minLogTe)/(dlogTe)+1;
    cell_center = (t_idx - 0.5)*dlogTe + minLogTe;
    cell_vals_2d[0] = 2.0*(log_Te_av - cell_center)/dlogTe; // Te value on cell interval
      
    if (log_m0_av < minLogM0) {
      m0_idx=1;
      log_m0_av = minLogM0;
    }
    else if (log_m0_av > maxLogM0) {
      m0_idx=resM0;
      log_m0_av = maxLogM0;
    }
    else m0_idx = (log_m0_av - minLogM0)/(dlogM0)+1;
    cell_center = (m0_idx - 0.5)*dlogM0 + minLogM0;
    cell_vals_2d[1] = 2.0*(log_m0_av - cell_center)/dlogM0; // M0 value on cell interval

    int ad_idx[2] = {t_idx, m0_idx};

    if ((m0_elc_av <= 0.) || (temp_elc_av <= 0.)) {
      coef_recomb_d[0] = 0.0;
    }   
    else {
      double *recomb_dat_d = (double*) gkyl_array_fetch(recomb_data, gkyl_range_idx(&adas_rng,ad_idx));
      double adas_eval = adas_basis->eval_expand(cell_vals_2d, recomb_dat_d);
      coef_recomb_d[0] = pow(10.0,adas_eval)/cell_av_fac;
    }
  }
}

void gkyl_dg_recomb_coll_cu(const struct gkyl_dg_recomb *up,
  const struct gkyl_array *prim_vars_elc, 
  struct gkyl_array *coef_recomb, struct gkyl_array *cflrate)
{  
  gkyl_recomb_react_rate_cu_ker<<<up->conf_rng->nblocks, up->conf_rng->nthreads>>>(up->on_dev, 
    *up->conf_rng, up->adas_rng, up->basis_on_dev,
    prim_vars_elc->on_dev, coef_recomb->on_dev, 
    up->recomb_data->on_dev, up->mass_elc, up->elem_charge, 
    up->maxLogTe, up->minLogTe, up->dlogTe, up->resTe, 
    up->maxLogM0, up->minLogM0, up->dlogM0, up->resM0);

  // cfl calculation
  //struct gkyl_range vel_rng;
  /* gkyl_range_deflate(&vel_rng, up->phase_rng, rem_dir, conf_iter.idx); */
  /* gkyl_range_iter_no_split_init(&vel_iter, &vel_rng); */
  /* // cfl associated with reaction is a *phase space* cfl */
  /* // Need to loop over velocity space for each configuration space cell */
  /* // to get total cfl rate in each phase space cell */
  /* while (gkyl_range_iter_next(&vel_iter)) { */
  /*   long cfl_idx = gkyl_range_idx(&vel_rng, vel_iter.idx); */
  /*   double *cflrate_d = gkyl_array_fetch(cflrate, cfl_idx); */
  /*   cflrate_d[0] += cflr; // frequencies are additive */
  /* } */
}
