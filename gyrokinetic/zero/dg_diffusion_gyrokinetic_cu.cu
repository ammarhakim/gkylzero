#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_diffusion_gyrokinetic.h>    
#include <gkyl_dg_diffusion_gyrokinetic_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_dg_diffusion_gyrokinetic_set_auxfields_cu_kernel(const struct gkyl_dg_eqn* eqn, const struct gkyl_array* D, const struct gkyl_array* jacobgeo_inv)
{
  struct dg_diffusion_gyrokinetic* diffusion = container_of(eqn, struct dg_diffusion_gyrokinetic, eqn);
  diffusion->auxfields.D = D;
  diffusion->auxfields.jacobgeo_inv = jacobgeo_inv;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_dg_diffusion_gyrokinetic_set_auxfields_cu(const struct gkyl_dg_eqn* eqn, struct gkyl_dg_diffusion_gyrokinetic_auxfields auxin)
{
  gkyl_dg_diffusion_gyrokinetic_set_auxfields_cu_kernel<<<1,1>>>(eqn, auxin.D->on_dev, auxin.jacobgeo_inv->on_dev);
}

__global__ void static
dg_diffusion_gyrokinetic_set_cu_dev_ptrs(struct dg_diffusion_gyrokinetic *diffusion, enum gkyl_basis_type b_type, int cdim, int vdim, int poly_order, int diff_order, int diffdirs_linidx)
{
  diffusion->auxfields.D = 0; 
  diffusion->auxfields.jacobgeo_inv = 0; 

  const gkyl_dg_diffusion_gyrokinetic_vol_kern_list *vol_kernels;
  const gkyl_dg_diffusion_gyrokinetic_surf_kern_list *surfx_kernels;
  const gkyl_dg_diffusion_gyrokinetic_surf_kern_list *surfy_kernels;
  const gkyl_dg_diffusion_gyrokinetic_surf_kern_list *surfz_kernels;
  const gkyl_dg_diffusion_gyrokinetic_boundary_surf_kern_list *boundary_surfx_kernels;
  const gkyl_dg_diffusion_gyrokinetic_boundary_surf_kern_list *boundary_surfy_kernels;
  const gkyl_dg_diffusion_gyrokinetic_boundary_surf_kern_list *boundary_surfz_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels            = diffusion->const_coeff? ser_vol_kernels_constcoeff                        : ser_vol_kernels_varcoeff                       ;
      surfx_kernels          = diffusion->const_coeff? ser_gyrokinetic_surfx_kernels_constcoeff          : ser_gyrokinetic_surfx_kernels_varcoeff         ;
      surfy_kernels          = diffusion->const_coeff? ser_gyrokinetic_surfy_kernels_constcoeff          : ser_gyrokinetic_surfy_kernels_varcoeff         ;
      surfz_kernels          = diffusion->const_coeff? ser_gyrokinetic_surfz_kernels_constcoeff          : ser_gyrokinetic_surfz_kernels_varcoeff         ;
      boundary_surfx_kernels = diffusion->const_coeff? ser_gyrokinetic_boundary_surfx_kernels_constcoeff : ser_gyrokinetic_boundary_surfx_kernels_varcoeff;
      boundary_surfy_kernels = diffusion->const_coeff? ser_gyrokinetic_boundary_surfy_kernels_constcoeff : ser_gyrokinetic_boundary_surfy_kernels_varcoeff;
      boundary_surfz_kernels = diffusion->const_coeff? ser_gyrokinetic_boundary_surfz_kernels_constcoeff : ser_gyrokinetic_boundary_surfz_kernels_varcoeff;
      break;

    default:
      assert(false);
      break;
  }

  diffusion->eqn.num_equations = 1;
  diffusion->eqn.surf_term = surf;
  diffusion->eqn.boundary_surf_term = boundary_surf;

  diffusion->eqn.vol_term = CKVOL(vol_kernels, cdim, diff_order, poly_order, diffdirs_linidx);

  diffusion->surf[0] = CKSURF(surfx_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>1)
    diffusion->surf[1] = CKSURF(surfy_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>2)
    diffusion->surf[2] = CKSURF(surfz_kernels, diff_order, cdim, vdim, poly_order);

  diffusion->boundary_surf[0] = CKSURF(boundary_surfx_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>1)
    diffusion->boundary_surf[1] = CKSURF(boundary_surfy_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>2)
    diffusion->boundary_surf[2] = CKSURF(boundary_surfz_kernels, diff_order, cdim, vdim, poly_order);
}

struct gkyl_dg_eqn*
gkyl_dg_diffusion_gyrokinetic_cu_dev_new(const struct gkyl_basis *basis, const struct gkyl_basis *cbasis,
  bool is_diff_const, const bool *diff_in_dir, int diff_order, const struct gkyl_range *diff_range, double skip_cell_threshold)
{
  struct dg_diffusion_gyrokinetic* diffusion = (struct dg_diffusion_gyrokinetic*) gkyl_malloc(sizeof(struct dg_diffusion_gyrokinetic));

  int cdim = cbasis->ndim;
  int vdim = basis->ndim - cdim;
  int poly_order = cbasis->poly_order;

  if (skip_cell_threshold > 0.0)
    diffusion->skip_cell_thresh = skip_cell_threshold * pow(sqrt(2.0), cdim + vdim);
  else
    diffusion->skip_cell_thresh = -1.0;

  diffusion->const_coeff = is_diff_const;
  diffusion->num_basis = basis->num_basis;
  for (int d=0; d<cdim; d++) diffusion->diff_in_dir[d] = diff_in_dir[d];

  int dirs_linidx = diffdirs_linidx(diff_in_dir, cdim);

  diffusion->diff_range = *diff_range;

  diffusion->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(diffusion->eqn.flags);
  diffusion->eqn.ref_count = gkyl_ref_count_init(gkyl_dg_diffusion_gyrokinetic_free);

  // copy the host struct to device struct
  struct dg_diffusion_gyrokinetic* diffusion_cu = (struct dg_diffusion_gyrokinetic*) gkyl_cu_malloc(sizeof(struct dg_diffusion_gyrokinetic));
  gkyl_cu_memcpy(diffusion_cu, diffusion, sizeof(struct dg_diffusion_gyrokinetic), GKYL_CU_MEMCPY_H2D);
  dg_diffusion_gyrokinetic_set_cu_dev_ptrs<<<1,1>>>(diffusion_cu, cbasis->b_type, cdim, vdim, poly_order, diff_order, dirs_linidx);

  // set parent on_dev pointer
  diffusion->eqn.on_dev = &diffusion_cu->eqn;

  return &diffusion->eqn;
}
