#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <assert.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_util.h>
#include <gkyl_mom_bcorr_lbo_gyrokinetic.h>
#include <gkyl_mom_bcorr_lbo_gyrokinetic_priv.h>
}

__global__
static void
gkyl_mom_bcorr_lbo_gyrokinetic_set_cu_dev_ptrs(struct mom_type_bcorr_lbo_gyrokinetic* mom_bcorr, enum gkyl_basis_type b_type, int vdim, int poly_order, int tblidx)
{
  mom_bcorr->momt.kernel = kernel;

  // choose kernel tables based on basis-function type
  const gkyl_mom_bcorr_lbo_gyrokinetic_kern_list *mom_bcorr_lbo_gyrokinetic_kernels;

  switch (b_type) {
  case GKYL_BASIS_MODAL_SERENDIPITY:
    mom_bcorr_lbo_gyrokinetic_kernels = ser_mom_bcorr_lbo_gyrokinetic_kernels;
    break;

  // case GKYL_BASIS_MODAL_TENSOR:
  //   mom_bcorr_lbo_gyrokinetic_kernels = ten_mom_bcorr_lbo_gyrokinetic_kernels;
  //   break;

  default:
    assert(false);
    break;
  }
  mom_bcorr->kernel = mom_bcorr_lbo_gyrokinetic_kernels[tblidx].kernels[poly_order];
  mom_bcorr->momt.num_mom = 2;
}

struct gkyl_mom_type*
gkyl_mom_bcorr_lbo_gyrokinetic_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis, 
  double mass, const struct gkyl_velocity_map *vel_map)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct mom_type_bcorr_lbo_gyrokinetic *mom_bcorr = (struct mom_type_bcorr_lbo_gyrokinetic*) gkyl_malloc(sizeof(*mom_bcorr));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  mom_bcorr->momt.cdim = cdim;
  mom_bcorr->momt.pdim = pdim;
  mom_bcorr->momt.poly_order = poly_order;
  mom_bcorr->momt.num_config = cbasis->num_basis;
  mom_bcorr->momt.num_phase = pbasis->num_basis;
  mom_bcorr->momt.num_mom = 2; // number of moments

  mom_bcorr->_m = mass;
  // Acquire pointers to on_dev objects so memcpy below copies those too.
  struct gkyl_velocity_map *vel_map_ho = gkyl_velocity_map_acquire(vel_map);
  mom_bcorr->vel_map = vel_map_ho->on_dev;

  mom_bcorr->momt.flags = 0;
  GKYL_SET_CU_ALLOC(mom_bcorr->momt.flags);
  mom_bcorr->momt.ref_count = gkyl_ref_count_init(gk_mom_free);

  // copy struct to device
  struct mom_type_bcorr_lbo_gyrokinetic *mom_bcorr_cu = (struct mom_type_bcorr_lbo_gyrokinetic*)
    gkyl_cu_malloc(sizeof(struct mom_type_bcorr_lbo_gyrokinetic));
  gkyl_cu_memcpy(mom_bcorr_cu, mom_bcorr, sizeof(struct mom_type_bcorr_lbo_gyrokinetic), GKYL_CU_MEMCPY_H2D);

  assert(cv_index[cdim].vdim[vdim] != -1);

  gkyl_mom_bcorr_lbo_gyrokinetic_set_cu_dev_ptrs<<<1,1>>>(mom_bcorr_cu, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  mom_bcorr->momt.on_dev = &mom_bcorr_cu->momt;

  // Updater should store host pointers.
  mom_bcorr->vel_map = vel_map_ho;

  return &mom_bcorr->momt;
}
