#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_fem_poisson.h>
#include <gkyl_fem_poisson_priv.h>
}

// CUDA kernel to set device pointers to l2g kernel function.
// Doing function pointer stuff in here avoids troublesome
// hipMemcpyFromSymbol.
__global__ static void
fem_poisson_set_cu_l2gker_ptrs(struct gkyl_fem_poisson_kernels* kers, enum gkyl_basis_type b_type,
  int dim, int poly_order, const int *bckey)
{

  // Set l2g kernels.
  const local2global_kern_bcx_list_1x *local2global_1x_kernels;
  const local2global_kern_bcx_list_2x *local2global_2x_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      local2global_1x_kernels = ser_loc2glob_list_1x;
      local2global_2x_kernels = ser_loc2glob_list_2x;
      break;
    default:
      assert(false);
      break;
  }

  for (int k=0; k<(int)(pow(2,dim)+0.5); k++) {
    if (dim == 1) {
      kers->l2g[k] = CK1(local2global_1x_kernels, poly_order, k, bckey[0]);
    } else if ( dim == 2) {
      kers->l2g[k] = CK2(local2global_2x_kernels, poly_order, k, bckey[0], bckey[1]);
//    } else if (dim == 3) {
//      kers->l2g[k] = CK3(ser_loc2glob_list_3x, poly_order, k, bckey[0], bckey[1], bckey[2]);
    }
  }

}

// CUDA kernel to set device pointers to RHS src and solution kernels.
__global__ static void
fem_poisson_set_cu_ker_ptrs(struct gkyl_fem_poisson_kernels* kers, enum gkyl_basis_type b_type,
  int dim, int poly_order, const int *bckey, bool isvareps)
{

  // Set RHS stencil kernels.
  const srcstencil_kern_bcx_list_1x *srcstencil_1x_kernels;
  const srcstencil_kern_bcx_list_2x *srcstencil_2x_kernels;

  if (isvareps) {
    switch (b_type) {
      case GKYL_BASIS_MODAL_SERENDIPITY:
          srcstencil_1x_kernels = ser_srcstencil_vareps_list_1x;
          srcstencil_2x_kernels = ser_srcstencil_vareps_list_2x;
        break;
//      case GKYL_BASIS_MODAL_TENSOR:
//        break;
      default:
        assert(false);
    }
  } else {
    switch (b_type) {
      case GKYL_BASIS_MODAL_SERENDIPITY:
          srcstencil_1x_kernels = ser_srcstencil_consteps_list_1x;
          srcstencil_2x_kernels = ser_srcstencil_consteps_list_2x;
        break;
//      case GKYL_BASIS_MODAL_TENSOR:
//        break;
      default:
        assert(false);
    }
  }

  for (int k=0; k<(int)(pow(3,dim)+0.5); k++) {
    if (dim == 1) {
      kers->srcker[k] = CK1(srcstencil_1x_kernels, poly_order, k, bckey[0]);
    } else if (dim == 2) {
      kers->srcker[k] = CK2(srcstencil_2x_kernels, poly_order, k, bckey[0], bckey[1]);
//  } else if (dim == 3) {
//    kers->srcker[k] = CK3(srcstencil_3x_kernels, poly_order, k, bckey[0], bckey[1], bckey[2]);
    }
  }

  // Set the get solution stencil kernel.
  const solstencil_kern_list *solstencil_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
        solstencil_kernels = ser_solstencil_list;
      break;
    case GKYL_BASIS_MODAL_TENSOR:
      break;
    default:
      assert(false);
  }

  kers->solker = solstencil_kernels[dim].kernels[poly_order];

}

// CUDA kernel to set device pointers to biasing kernel functions.
// Doing function pointer stuff in here avoids troublesome
// hipMemcpyFromSymbol.
__global__ static void
fem_poisson_set_cu_biasker_ptrs(struct gkyl_fem_poisson_kernels* kers, enum gkyl_basis_type b_type,
  int dim, int poly_order, const int *bckey)
{

  // Set l2g kernels.
  const bias_src_kern_bcx_list_1x *bias_plane_1x_kernels;
  const bias_src_kern_bcx_list_2x *bias_plane_2x_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      bias_plane_1x_kernels = ser_bias_src_list_1x;
      bias_plane_2x_kernels = ser_bias_src_list_2x;
      break;
    default:
      assert(false);
      break;
  }

  for (int k=0; k<(int)(pow(2,dim)+0.5); k++) {
    if (dim == 1) {
      kers->bias_src_ker[k] = CK1(bias_plane_1x_kernels, poly_order, k, bckey[0]);
    } else if ( dim == 2) {
      kers->bias_src_ker[k] = CK2(bias_plane_2x_kernels, poly_order, k, bckey[0], bckey[1]);
//    } else if (dim == 3) {
//      kers->bias_src_ker[k] = CK3(bias_plane_3x_kernels, poly_order, k, bckey[0], bckey[1], bckey[2]);
    }
  }

}

void
fem_poisson_choose_kernels_cu(const struct gkyl_basis* basis, const struct gkyl_poisson_bc *bcs,
  bool isvareps, const bool *isdirperiodic, struct gkyl_fem_poisson_kernels *kers)
{

  int dim = basis->ndim;
  int poly_order = basis->poly_order;

  int bckey[GKYL_MAX_CDIM] = {-1};
  for (int d=0; d<basis->ndim; d++) bckey[d] = isdirperiodic[d] ? 0 : 1;
  int *bckey_d = (int *) gkyl_cu_malloc(sizeof(int[GKYL_MAX_CDIM]));
  gkyl_cu_memcpy(bckey_d, bckey, sizeof(int[GKYL_MAX_CDIM]), GKYL_CU_MEMCPY_H2D);

  fem_poisson_set_cu_l2gker_ptrs<<<1,1>>>(kers, basis->b_type, dim, poly_order, bckey_d);
  
  // Biasing kernels (set this before redefining bckey_d below).
  fem_poisson_set_cu_biasker_ptrs<<<1,1>>>(kers, basis->b_type, dim, poly_order, bckey_d);

  for (int d=0; d<basis->ndim; d++) {
         if (bcs->lo_type[d]==GKYL_POISSON_PERIODIC          && bcs->up_type[d]==GKYL_POISSON_PERIODIC         ) { bckey[d] = 0; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET         && bcs->up_type[d]==GKYL_POISSON_DIRICHLET        ) { bckey[d] = 1; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET         && bcs->up_type[d]==GKYL_POISSON_NEUMANN          ) { bckey[d] = 2; }
    else if (bcs->lo_type[d]==GKYL_POISSON_NEUMANN           && bcs->up_type[d]==GKYL_POISSON_DIRICHLET        ) { bckey[d] = 3; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET         && bcs->up_type[d]==GKYL_POISSON_ROBIN            ) { bckey[d] = 4; }
    else if (bcs->lo_type[d]==GKYL_POISSON_ROBIN             && bcs->up_type[d]==GKYL_POISSON_DIRICHLET        ) { bckey[d] = 5; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET         && bcs->up_type[d]==GKYL_POISSON_DIRICHLET_VARYING) { bckey[d] = 6; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET_VARYING && bcs->up_type[d]==GKYL_POISSON_DIRICHLET        ) { bckey[d] = 7; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET_VARYING && bcs->up_type[d]==GKYL_POISSON_DIRICHLET_VARYING) { bckey[d] = 8; }
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET_VARYING && bcs->up_type[d]==GKYL_POISSON_NEUMANN          ) { bckey[d] = 9; }
    else if (bcs->lo_type[d]==GKYL_POISSON_NEUMANN           && bcs->up_type[d]==GKYL_POISSON_DIRICHLET_VARYING) { bckey[d] = 10; }
    // MF 2024/10/01: kernels for these two are not yet plugged into the big lists above.
    else if (bcs->lo_type[d]==GKYL_POISSON_DIRICHLET_VARYING && bcs->up_type[d]==GKYL_POISSON_ROBIN            ) { bckey[d] = 11; }
    else if (bcs->lo_type[d]==GKYL_POISSON_ROBIN             && bcs->up_type[d]==GKYL_POISSON_DIRICHLET_VARYING) { bckey[d] = 12; }
    else { assert(false); }
  };
  gkyl_cu_memcpy(bckey_d, bckey, sizeof(int[GKYL_MAX_CDIM]), GKYL_CU_MEMCPY_H2D);

  fem_poisson_set_cu_ker_ptrs<<<1,1>>>(kers, basis->b_type, dim, poly_order, bckey_d, isvareps);

  gkyl_cu_free(bckey_d);
}

__global__ void
gkyl_fem_poisson_set_rhs_kernel(struct gkyl_array *epsilon, bool isvareps, const double *dx, double *rhs_global,
  struct gkyl_array *rhs_local, struct gkyl_range range, const double *bcvals, const struct gkyl_array *phibc,
  struct gkyl_fem_poisson_kernels *kers)
{
  int idx[GKYL_MAX_CDIM];  int idx0[GKYL_MAX_CDIM];  int num_cells[GKYL_MAX_CDIM];
  long globalidx[32];
  for (int d=0; d<GKYL_MAX_CDIM; d++) num_cells[d] = range.upper[d]-range.lower[d]+1;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
       linc1 < range.volume;
       linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);
    
    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);
    
    const double *local_d = (const double*) gkyl_array_cfetch(rhs_local, start);
    const double *epsilon_d = isvareps? (const double*) gkyl_array_cfetch(epsilon, start)
                                      : (const double*) gkyl_array_cfetch(epsilon, 0);
    const double *phibc_d = phibc? (const double *) gkyl_array_cfetch(phibc, start) : NULL;

    int keri = idx_to_inup_ker(range.ndim, num_cells, idx);
    for (size_t d=0; d<range.ndim; d++) idx0[d] = idx[d]-1;
    kers->l2g[keri](num_cells, idx0, globalidx);

    // Apply the RHS source stencil. It's mostly the mass matrix times a
    // modal-to-nodal operator times the source, modified by BCs in skin cells.
    keri = idx_to_inloup_ker(range.ndim, num_cells, idx);
    kers->srcker[keri](epsilon_d, dx, local_d, bcvals, phibc_d, globalidx, rhs_global);
  }
}

__global__ void
gkyl_fem_poisson_get_sol_kernel(struct gkyl_array *x_local, const double *x_global,
  struct gkyl_range range, struct gkyl_fem_poisson_kernels *kers)
{
  int idx[GKYL_MAX_CDIM];  int idx0[GKYL_MAX_CDIM];  int num_cells[GKYL_MAX_CDIM];
  long globalidx[32];
  for (int d=0; d<GKYL_MAX_CDIM; d++) num_cells[d] = range.upper[d]-range.lower[d]+1;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
       linc1 < range.volume;
       linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);
    
    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);
    
    double *local_d = (double*) gkyl_array_cfetch(x_local, start);

    int keri = idx_to_inup_ker(range.ndim, num_cells, idx);
    for (size_t d=0; d<range.ndim; d++) idx0[d] = idx[d]-1;
    kers->l2g[keri](num_cells, idx0, globalidx);

    // Apply the RHS source stencil. It's mostly the mass matrix times a
    // modal-to-nodal operator times the source, modified by BCs in skin cells.
    kers->solker(x_global, globalidx, local_d);
  }
}

__global__ void
gkyl_fem_poisson_bias_src_kernel(double *rhs_global, struct gkyl_rect_grid grid,
  struct gkyl_range range, struct gkyl_fem_poisson_kernels *kers,
  int num_bias_plane, struct gkyl_poisson_bias_plane *bias_planes)
{
  int idx[GKYL_MAX_CDIM];  int idx0[GKYL_MAX_CDIM];  int num_cells[GKYL_MAX_CDIM];
  long globalidx[32];
  for (int d=0; d<GKYL_MAX_CDIM; d++) num_cells[d] = range.upper[d]-range.lower[d]+1;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
       linc1 < range.volume;
       linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);
    
    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);
    
    int keri = idx_to_inup_ker(range.ndim, num_cells, idx);
    for (size_t d=0; d<range.ndim; d++) idx0[d] = idx[d]-1;
    kers->l2g[keri](num_cells, idx0, globalidx);

    // Apply the RHS source stencil. It's mostly the mass matrix times a
    // modal-to-nodal operator times the source, modified by BCs in skin cells.
    for (int i=0; i<num_bias_plane; i++) {
      // Index of the cell that abuts the plane from below.
      struct gkyl_poisson_bias_plane *bp = &bias_planes[i];
      double dx = grid.dx[bp->dir];
      int bp_idx_m = (bp->loc-1e-3*dx - grid.lower[bp->dir])/dx+1;

      if (idx[bp->dir] == bp_idx_m || idx[bp->dir] == bp_idx_m+1) {
        kers->bias_src_ker[keri](-1+2*((bp_idx_m+1)-idx[bp->dir]),
          bp->dir, bp->val, globalidx, rhs_global);
      }
    }
  }
}

void 
gkyl_fem_poisson_bias_src_cu(gkyl_fem_poisson *up, struct gkyl_array *rhsin)
{
  double *rhs_cu = gkyl_culinsolver_get_rhs_ptr(up->prob_cu, 0);
  gkyl_fem_poisson_bias_src_kernel<<<rhsin->nblocks, rhsin->nthreads>>>(rhs_cu, up->grid,
    *up->solve_range, up->kernels_cu, up->num_bias_plane, up->bias_planes); 
}	

void 
gkyl_fem_poisson_set_rhs_cu(gkyl_fem_poisson *up, struct gkyl_array *rhsin, const struct gkyl_array *phibc)
{
  gkyl_culinsolver_clear_rhs(up->prob_cu, 0);
  double *rhs_cu = gkyl_culinsolver_get_rhs_ptr(up->prob_cu, 0);
  const struct gkyl_array *phibc_cu = phibc? phibc->on_dev : NULL;
  gkyl_fem_poisson_set_rhs_kernel<<<rhsin->nblocks, rhsin->nthreads>>>(up->epsilon->on_dev,
    up->isvareps, up->dx_cu, rhs_cu, rhsin->on_dev, *up->solve_range, up->bcvals_cu,
    phibc_cu, up->kernels_cu); 

  // Set the corresponding entries to the biasing potential.
  up->bias_plane_src(up, rhsin);
}	

void
gkyl_fem_poisson_solve_cu(gkyl_fem_poisson *up, struct gkyl_array *phiout)
{
  // Do linear solve with cusolver.
  gkyl_culinsolver_solve(up->prob_cu);
  double *x_cu = gkyl_culinsolver_get_sol_ptr(up->prob_cu, 0);

  gkyl_fem_poisson_get_sol_kernel<<<phiout->nblocks, phiout->nthreads>>>(phiout->on_dev,
    x_cu, *up->solve_range, up->kernels_cu); 
}

