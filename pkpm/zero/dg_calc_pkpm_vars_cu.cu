#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_calc_pkpm_vars.h>
#include <gkyl_dg_calc_pkpm_vars_priv.h>
#include <gkyl_wave_geom.h>
#include <gkyl_wv_eqn.h>
#include <gkyl_util.h>
}

__global__ static void
gkyl_dg_calc_pkpm_vars_set_cu_kernel(gkyl_dg_calc_pkpm_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* pkpm_div_ppar, 
  struct gkyl_array* cell_avg_prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const double *pkpm_div_ppar_d = (const double*) gkyl_array_cfetch(pkpm_div_ppar, loc);

    int* cell_avg_prim_d = (int*) gkyl_array_fetch(cell_avg_prim, loc);

    cell_avg_prim_d[0] = up->pkpm_set(count, As, xs, 
      vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, pkpm_div_ppar_d);
  }
}

__global__ static void
gkyl_dg_calc_pkpm_vars_copy_cu_kernel(gkyl_dg_calc_pkpm_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  struct gkyl_array* prim, struct gkyl_array* prim_surf)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    double* prim_d = (double*) gkyl_array_fetch(prim, loc);
    double* prim_surf_d = (double*) gkyl_array_fetch(prim_surf, loc);

    up->pkpm_copy(count, xs, prim_d, prim_surf_d);
  }
}

// Host-side wrapper for pkpm primitive variable calculation
void gkyl_dg_calc_pkpm_vars_advance_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* pkpm_div_ppar, 
  struct gkyl_array* cell_avg_prim, struct gkyl_array* prim, struct gkyl_array* prim_surf)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_pkpm_vars_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As->on_dev, up->xs->on_dev, conf_range,
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    p_ij->on_dev, pkpm_div_ppar->on_dev, 
    cell_avg_prim->on_dev);

  if (up->poly_order > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem, up->As, up->xs);
    assert(status);
  }

  gkyl_dg_calc_pkpm_vars_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs->on_dev, conf_range, prim->on_dev, prim_surf->on_dev);
}

__global__ static void
gkyl_dg_calc_pkpm_vars_u_set_cu_kernel(gkyl_dg_calc_pkpm_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* cell_avg_prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*3;

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);

    int* cell_avg_prim_d = (int*) gkyl_array_fetch(cell_avg_prim, loc);

    cell_avg_prim_d[0] = up->pkpm_u_set(count, As, xs, 
      vlasov_pkpm_moms_d, euler_pkpm_d);
  }
}

__global__ static void
gkyl_dg_calc_pkpm_vars_u_copy_cu_kernel(gkyl_dg_calc_pkpm_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  struct gkyl_array* pkpm_u)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*3;

    double* pkpm_u_d = (double*) gkyl_array_fetch(pkpm_u, loc);

    up->pkpm_u_copy(count, xs, pkpm_u_d);
  }
}

// Host-side wrapper for pkpm flow velocity calculation
void gkyl_dg_calc_pkpm_vars_u_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* cell_avg_prim, struct gkyl_array* pkpm_u)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_pkpm_vars_u_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As_u->on_dev, up->xs_u->on_dev, conf_range,
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    cell_avg_prim->on_dev);

  if (up->poly_order > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem_u, up->As_u, up->xs_u);
    assert(status);
  }

  gkyl_dg_calc_pkpm_vars_u_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs_u->on_dev, conf_range, pkpm_u->on_dev);
}

__global__ void
gkyl_calc_pkpm_vars_pressure_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{ 
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *bvar_d = (const double*) gkyl_array_cfetch(bvar, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);

    double *p_ij_d = (double*) gkyl_array_fetch(p_ij, loc);
    up->pkpm_pressure(bvar_d, vlasov_pkpm_moms_d, p_ij_d);
  }
}

// Host-side wrapper for pkpm pressure calculation
void gkyl_dg_calc_pkpm_vars_pressure_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_calc_pkpm_vars_pressure_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    bvar->on_dev, vlasov_pkpm_moms->on_dev, p_ij->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_accel_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* prim_surf, const struct gkyl_array* prim, 
  const struct gkyl_array* bvar, const struct gkyl_array* div_b, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_accel)
{
  int cdim = up->cdim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idxc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&conf_range, idxc);

    const double *prim_surf_c = (const double*) gkyl_array_cfetch(prim_surf, linc);

    const double *prim_d = (const double*) gkyl_array_cfetch(prim, linc);
    const double *bvar_d = (const double*) gkyl_array_cfetch(bvar, linc);
    const double *div_b_d = (const double*) gkyl_array_cfetch(div_b, linc);
    const double *nu_d = (const double*) gkyl_array_cfetch(nu, linc);

    double *pkpm_accel_d = (double*) gkyl_array_fetch(pkpm_accel, linc);

    // Compute T_perp/m div(b) and p_force
    up->pkpm_p_force(prim_d, div_b_d, pkpm_accel_d);

    for (int dir=0; dir<cdim; ++dir) {
      gkyl_copy_int_arr(cdim, idxc, idxl);
      gkyl_copy_int_arr(cdim, idxc, idxr);

      idxl[dir] = idxl[dir]-1; idxr[dir] = idxr[dir]+1;

      long linl = gkyl_range_idx(&conf_range, idxl); 
      long linr = gkyl_range_idx(&conf_range, idxr);

      const double *prim_surf_l = (const double*) gkyl_array_cfetch(prim_surf, linl);
      const double *prim_surf_r = (const double*) gkyl_array_cfetch(prim_surf, linr);
      
      up->pkpm_accel[dir](up->conf_grid.dx, 
        prim_surf_l, prim_surf_c, prim_surf_r, 
        prim_d, bvar_d, nu_d,
        pkpm_accel_d);
    }
  }
}

// Host-side wrapper for pkpm acceleration variable calculations with averaging for gradients
void
gkyl_dg_calc_pkpm_vars_accel_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* prim_surf, const struct gkyl_array* prim, 
  const struct gkyl_array* bvar, const struct gkyl_array* div_b, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_accel)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_accel_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    prim_surf->on_dev, prim->on_dev, 
    bvar->on_dev, div_b->on_dev, nu->on_dev, 
    pkpm_accel->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_penalization_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range conf_range_ext, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* pkpm_lax, struct gkyl_array* pkpm_penalization)
{
  int cdim = up->cdim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idxc);

    const struct gkyl_wave_cell_geom *geom = gkyl_wave_geom_get(up->geom, idxc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&conf_range, idxc);
  
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, linc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, linc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, linc);

    double *pkpm_lax_d = (double*) gkyl_array_fetch(pkpm_lax, linc);
    double *pkpm_penalization_d = (double*) gkyl_array_fetch(pkpm_penalization, linc);

    for (int dir=0; dir<cdim; ++dir) {
      gkyl_copy_int_arr(cdim, idxc, idxl);

      // Each cell owns their *lower* edge surface evaluation so we need both 
      // the cell we are in (linc) and our lower neighbor (linl) to compute 
      // the penalization surface expansions at the lower edge surface. 
      idxl[dir] = idxl[dir]-1; 
      long linl = gkyl_range_idx(&conf_range, idxl); 

      const double *vlasov_pkpm_moms_l = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linl);
      const double *p_ij_l = (const double*) gkyl_array_cfetch(p_ij, linl);
      const double *prim_l = (const double*) gkyl_array_cfetch(prim, linl);
      const double *euler_pkpm_l = (const double*) gkyl_array_cfetch(euler_pkpm, linl);

      up->pkpm_penalization[dir](up->tol, up->force_lax, up->wv_eqn, geom, 
        vlasov_pkpm_moms_l, vlasov_pkpm_moms_d, p_ij_l, p_ij_d, 
        prim_l, prim_d, euler_pkpm_l, euler_pkpm_d, 
        pkpm_lax_d, pkpm_penalization_d);

      // If the configuration-space index is at the local configuration space upper value, 
      // we are at the configuration space upper edge and we also need to evaluate the 
      // penalization terms at the upper edge interface. We index into the ghost cells (linr)
      // and following the convention of each cell owning their lower surface expansion,
      // the upper edge surface expansions are store in the ghost cells of the array. 
      if (idxc[dir] == conf_range.upper[dir]) {
        gkyl_copy_int_arr(cdim, idxc, idxr);
        idxr[dir] = idxr[dir]+1; 
        long linr = gkyl_range_idx(&conf_range_ext, idxr);

        const struct gkyl_wave_cell_geom *geom_r = gkyl_wave_geom_get(up->geom, idxr);

        const double *vlasov_pkpm_moms_r = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linr);
        const double *p_ij_r = (const double*) gkyl_array_cfetch(p_ij, linr);
        const double *prim_r = (const double*) gkyl_array_cfetch(prim, linr);
        const double *euler_pkpm_r = (const double*) gkyl_array_cfetch(euler_pkpm, linr);

        double *pkpm_lax_r = (double*) gkyl_array_fetch(pkpm_lax, linr);
        double *pkpm_penalization_r = (double*) gkyl_array_fetch(pkpm_penalization, linr);

        up->pkpm_penalization[dir](up->tol, up->force_lax, up->wv_eqn, geom_r, 
          vlasov_pkpm_moms_d, vlasov_pkpm_moms_r, p_ij_d, p_ij_r, 
          prim_d, prim_r, euler_pkpm_d, euler_pkpm_r, 
          pkpm_lax_r, pkpm_penalization_r);
      }
    }
  }
}

// Host-side wrapper for surface expansions of pkpm penalization variables calculation
void
gkyl_dg_calc_pkpm_vars_penalization_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_range *conf_range_ext, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* pkpm_lax, struct gkyl_array* pkpm_penalization)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_penalization_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *conf_range_ext, 
    vlasov_pkpm_moms->on_dev, p_ij->on_dev, prim->on_dev, euler_pkpm->on_dev, 
    pkpm_lax->on_dev, pkpm_penalization->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_integrated_vars_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);

    double *int_pkpm_vars_d = (double*) gkyl_array_fetch(int_pkpm_vars, loc);
    up->pkpm_int(vlasov_pkpm_moms_d, euler_pkpm_d, prim_d, int_pkpm_vars_d);
  }
}

// Host-side wrapper for pkpm integrated variables calculation
void
gkyl_dg_calc_pkpm_integrated_vars_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_integrated_vars_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, prim->on_dev, 
    int_pkpm_vars->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_source_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm,
  struct gkyl_array* rhs)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *qmem_d = (const double*) gkyl_array_cfetch(qmem, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);

    double *rhs_d = (double*) gkyl_array_fetch(rhs, loc);
    up->pkpm_source(qmem_d, vlasov_pkpm_moms_d, euler_pkpm_d, rhs_d);
  }
}

// Host-side wrapper for pkpm source term calculations
void
gkyl_dg_calc_pkpm_vars_source_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* rhs)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_source_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    qmem->on_dev, vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    rhs->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_io_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);
    const double *pkpm_accel_d = (const double*) gkyl_array_cfetch(pkpm_accel, loc);

    double *fluid_io_d = (double*) gkyl_array_fetch(fluid_io, loc);
    double *pkpm_vars_io_d = (double*) gkyl_array_fetch(pkpm_vars_io, loc);
    up->pkpm_io(vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, prim_d, pkpm_accel_d, 
      fluid_io_d, pkpm_vars_io_d);
  }
}

// Host-side wrapper for pkpm io. Computes conserved variables and copies primitive and acceleration variables to output array
void
gkyl_dg_calc_pkpm_vars_io_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_io_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, p_ij->on_dev, prim->on_dev, pkpm_accel->on_dev, 
    fluid_io->on_dev, pkpm_vars_io->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_limiter_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, 
  struct gkyl_range conf_range, const struct gkyl_array* prim, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* p_ij, 
  struct gkyl_array* fluid)
{
  int cdim = up->cdim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idxc);
    const struct gkyl_wave_cell_geom *geom = gkyl_wave_geom_get(up->geom, idxc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&conf_range, idxc);

    const double *prim_c = (const double*) gkyl_array_cfetch(prim, linc);
    const double *vlasov_pkpm_moms_c = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linc);
    const double *p_ij_c = (const double*) gkyl_array_cfetch(p_ij, linc);

    double *fluid_c = (double*) gkyl_array_fetch(fluid, linc);
    for (int dir=0; dir<cdim; ++dir) {
      gkyl_copy_int_arr(cdim, idxc, idxl);
      gkyl_copy_int_arr(cdim, idxc, idxr);

      idxl[dir] = idxl[dir]-1; idxr[dir] = idxr[dir]+1;

      long linl = gkyl_range_idx(&conf_range, idxl); 
      long linr = gkyl_range_idx(&conf_range, idxr);

      const double *vlasov_pkpm_moms_l = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linl);
      const double *vlasov_pkpm_moms_r = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, linr);
      const double *p_ij_l = (const double*) gkyl_array_cfetch(p_ij, linl);
      const double *p_ij_r = (const double*) gkyl_array_cfetch(p_ij, linr);

      double *fluid_l = (double*) gkyl_array_fetch(fluid, linl);
      double *fluid_r = (double*) gkyl_array_fetch(fluid, linr);

      up->pkpm_limiter[dir](up->limiter_fac, up->wv_eqn, geom, prim_c, 
        vlasov_pkpm_moms_l, vlasov_pkpm_moms_c, vlasov_pkpm_moms_r, 
        p_ij_l, p_ij_c, p_ij_r, 
        fluid_l, fluid_c, fluid_r); 
    }
  }
}

// Host-side wrapper for slope limiter of fluid variables
void
gkyl_dg_calc_pkpm_vars_limiter_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_array* prim, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* p_ij, 
  struct gkyl_array* fluid)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_limiter_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    prim->on_dev, vlasov_pkpm_moms->on_dev, p_ij->on_dev, 
    fluid->on_dev);
}

// CUDA kernel to set device pointers to pkpm vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_pkpm_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_pkpm_vars *up, enum gkyl_basis_type b_type,
  int cdim, int poly_order)
{
  up->pkpm_set = choose_pkpm_set_kern(b_type, cdim, poly_order);
  up->pkpm_copy = choose_pkpm_copy_kern(b_type, cdim, poly_order);
  up->pkpm_u_set = choose_pkpm_u_set_kern(b_type, cdim, poly_order);
  up->pkpm_u_copy = choose_pkpm_u_copy_kern(b_type, cdim, poly_order);  
  up->pkpm_pressure = choose_pkpm_pressure_kern(b_type, cdim, poly_order);
  up->pkpm_p_force = choose_pkpm_p_force_kern(b_type, cdim, poly_order);
  up->pkpm_source = choose_pkpm_source_kern(b_type, cdim, poly_order);
  up->pkpm_int = choose_pkpm_int_kern(b_type, cdim, poly_order);
  up->pkpm_io = choose_pkpm_io_kern(b_type, cdim, poly_order);
  // Fetch the kernels in each direction
  for (int d=0; d<cdim; ++d) {
    up->pkpm_accel[d] = choose_pkpm_accel_kern(d, b_type, cdim, poly_order);
    up->pkpm_penalization[d] = choose_pkpm_penalization_kern(d, b_type, cdim, poly_order);
    up->pkpm_limiter[d] = choose_pkpm_limiter_kern(d, b_type, cdim, poly_order);
  }
}

gkyl_dg_calc_pkpm_vars*
gkyl_dg_calc_pkpm_vars_cu_dev_new(const struct gkyl_rect_grid *conf_grid, 
  const struct gkyl_basis* cbasis, const struct gkyl_range *mem_range, 
  const struct gkyl_wv_eqn *wv_eqn, const struct gkyl_wave_geom *wg, double limiter_fac)
{
  struct gkyl_dg_calc_pkpm_vars *up = (struct gkyl_dg_calc_pkpm_vars*) gkyl_malloc(sizeof(gkyl_dg_calc_pkpm_vars));

  up->conf_grid = *conf_grid;
  int nc = cbasis->num_basis;
  int cdim = cbasis->ndim;
  int poly_order = cbasis->poly_order;
  enum gkyl_basis_type b_type = cbasis->b_type;
  up->cdim = cdim;
  up->poly_order = poly_order;
  up->Ncomp = 9;
  up->mem_range = *mem_range;

  // acquire pointer to wave equation object
  struct gkyl_wv_eqn *eqn = gkyl_wv_eqn_acquire(wv_eqn);
  up->wv_eqn = eqn->on_dev; // this is so the memcpy below has eqn on_dev

  // acquire pointer to wave equation object
  struct gkyl_wave_geom *geom = gkyl_wave_geom_acquire(wg);
  up->geom = geom->on_dev; // this is so the memcpy below has geom on_dev

  // Limiter factor for relationship between slopes and cell average differences
  // By default, this factor is 1/sqrt(3) because cell_avg(f) = f0/sqrt(2^cdim)
  // and a cell slope estimate from two adjacent cells is (for the x variation): 
  // integral(psi_1 [cell_avg(f_{i+1}) - cell_avg(f_{i})]*x) = sqrt(2^cdim)/sqrt(3)*[cell_avg(f_{i+1}) - cell_avg(f_{i})]
  // where psi_1 is the x cell slope basis in our orthonormal expansion psi_1 = sqrt(3)/sqrt(2^cdim)*x
  // This factor can be made smaller (larger) to increase (decrease) the diffusion from the slope limiter
  if (limiter_fac == 0.0) {
    up->limiter_fac = 0.5773502691896258;
  }
  else {
    up->limiter_fac = limiter_fac;
  }

  // Tolerance in mass density and average normal velocity at the interface
  // for switching to Lax fluxes in computing penalization of the momentum solve
  up->tol = 1.0e-12;
  up->force_lax = false; 

  // There are Ncomp*range->volume linear systems to be solved 
  // 6 components: ux, uy, uz, div(p_par b)/rho, p_perp/rho, rho/p_perp
  up->As = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, nc);
  up->xs = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, 1);
  up->mem = gkyl_nmat_linsolve_lu_cu_dev_new(up->As->num, up->As->nr);

  // Linear system for just solving for ux, uy, uz
  up->As_u = gkyl_nmat_cu_dev_new(3*mem_range->volume, nc, nc);
  up->xs_u = gkyl_nmat_cu_dev_new(3*mem_range->volume, nc, 1);
  up->mem_u = gkyl_nmat_linsolve_lu_cu_dev_new(up->As_u->num, up->As_u->nr);

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_pkpm_vars *up_cu = (struct gkyl_dg_calc_pkpm_vars*) gkyl_cu_malloc(sizeof(gkyl_dg_calc_pkpm_vars));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_pkpm_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_pkpm_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, b_type, cdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;

  up->wv_eqn = eqn; // updater should store host pointer 
  up->geom = geom; 
  
  return up;
}
