#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_dg_calc_gyrokinetic_vars.h>
#include <gkyl_dg_calc_gyrokinetic_vars_priv.h>
#include <gkyl_util.h>
}

static void
gkyl_parallelize_components_kernel_launch_dims(dim3* dimGrid, dim3* dimBlock, gkyl_range range, int ncomp)
{
  // Create a 2D thread grid so we launch ncomp*range.volume number of threads and can parallelize over components too
  dimBlock->y = ncomp;
  dimGrid->y = 1;
  dimBlock->x = gkyl_int_div_up(252, ncomp);
  dimGrid->x = gkyl_int_div_up(range.volume, dimBlock->x);
}

__global__ void
gkyl_dg_calc_gyrokinetic_vars_flux_surf_cu_kernel(struct gkyl_dg_calc_gyrokinetic_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  struct gkyl_range conf_ext_range, struct gkyl_range phase_ext_range, const struct gkyl_array *phi, 
  const struct gkyl_array *fin, struct gkyl_array* flux_surf, struct gkyl_array *cflrate)
{ 
  int pdim = up->pdim;
  int cdim = up->cdim;
  int idx[GKYL_MAX_DIM], idx_edge[GKYL_MAX_DIM], idx_vel[2];
  int idxL[GKYL_MAX_DIM];
  int idx_velL[2];
  double xc[GKYL_MAX_DIM];

  // 2D thread grid
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    for (int d=cdim; d<pdim; d++) idx_vel[d-cdim] = idx[d];

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_vel = gkyl_range_idx(&up->vel_map->local_vel, idx_vel);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double *bmag_d = (const double*) gkyl_array_cfetch(up->gk_geom->geo_int.bmag, loc_conf);
    const double *phi_d = (const double*) gkyl_array_cfetch(phi, loc_conf);
    const double *vmap_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap, loc_vel);
    const double *vmapSq_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap_sq, loc_vel);

    double* flux_surf_d = (double*) gkyl_array_fetch(flux_surf, loc_phase);
    double *cflrate_d = (double*) gkyl_array_fetch(cflrate, loc_phase);

    for (int dir = 0; dir<cdim; ++dir) {
      gkyl_copy_int_arr(pdim, idx, idxL);
      idxL[dir] = idx[dir] - 1;
      long locL = gkyl_range_idx(&phase_range, idxL);
      const double *fL = (const double*) gkyl_array_cfetch(fin, locL);
      const double *fR = (const double*) gkyl_array_cfetch(fin, loc_phase);

      const struct gkyl_dg_surf_geom *dgs = gkyl_dg_geom_get_surf(up->dg_geom, dir, idx);
      const struct gkyl_gk_dg_surf_geom *gkdgs = gkyl_gk_dg_geom_get_surf(up->gk_dg_geom, dir, idx);
      cflrate_d[0] += up->flux_surf[dir](xc, up->phase_grid.dx, 
        vmap_d, vmapSq_d, up->charge, up->mass,
        dgs, gkdgs,
        bmag_d, phi_d,  fL, fR, flux_surf_d);

      // If the phase space index is at the local configuration space upper value, we
      // we are at the configuration space upper edge and we also need to evaluate 
      // alpha = +1 to avoid evaluating the geometry information in the ghost cells 
      // where it is not defined when computing the final surface alpha we need
      // (since the surface alpha array stores only the *lower* surface expansion)
      if (idx[dir] == phase_range.upper[dir]) {
        gkyl_copy_int_arr(pdim, idx, idx_edge);
        idx_edge[dir] = idx_edge[dir]+1;
        long loc_conf_ext = gkyl_range_idx(&conf_ext_range, idx_edge);
        long loc_phase_ext = gkyl_range_idx(&phase_ext_range, idx_edge);

        double *cflrate_ext_d = (double*) gkyl_array_fetch(cflrate, loc_phase_ext);
        const double *fL = (const double*)  gkyl_array_cfetch(fin, loc_phase);
        const double *fR = (const double*)  gkyl_array_cfetch(fin, loc_phase_ext);
        const struct gkyl_dg_surf_geom *dgs = gkyl_dg_geom_get_surf(up->dg_geom, dir, idx_edge);
        const struct gkyl_gk_dg_surf_geom *gkdgs = gkyl_gk_dg_geom_get_surf(up->gk_dg_geom, dir, idx_edge);

        double* flux_surf_ext_d = (double*) gkyl_array_fetch(flux_surf, loc_phase_ext);
        cflrate_ext_d[0] = up->flux_edge_surf[dir](xc, up->phase_grid.dx, 
          vmap_d, vmapSq_d, up->charge, up->mass,
          dgs, gkdgs,
          bmag_d, phi_d, fL, fR, flux_surf_ext_d);
      }  
    }
  }
}

__global__ void
gkyl_dg_calc_gyrokinetic_vars_flux_surfvpar_cu_kernel(struct gkyl_dg_calc_gyrokinetic_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range,
  struct gkyl_range conf_ext_range, struct gkyl_range phase_ext_range, struct gkyl_range vpar_range,
  const struct gkyl_array *phi, const struct gkyl_array *fin, 
  struct gkyl_array* flux_surf, struct gkyl_array *cflrate)
{ 
  int pdim = up->pdim;
  int cdim = up->cdim;
  int idx[GKYL_MAX_DIM], idx_edge[GKYL_MAX_DIM], idx_vel[2];
  int idxL[GKYL_MAX_DIM];
  int idx_velL[2];
  double xc[GKYL_MAX_DIM];

  // 2D thread grid
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < vpar_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&vpar_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    for (int d=cdim; d<pdim; d++) idx_vel[d-cdim] = idx[d];

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_vel = gkyl_range_idx(&up->vel_map->local_vel, idx_vel);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    const double *bmag_d = (const double*) gkyl_array_cfetch(up->gk_geom->geo_int.bmag, loc_conf);
    const double *phi_d = (const double*) gkyl_array_cfetch(phi, loc_conf);
    const double *vmap_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap, loc_vel);
    const double *vmapSq_d = (const double*) gkyl_array_cfetch(up->vel_map->vmap_sq, loc_vel);

    double* flux_surf_d = (double*) gkyl_array_fetch(flux_surf, loc_phase);
    double *cflrate_d = (double*) gkyl_array_fetch(cflrate, loc_phase);

    int dir = cdim;
    gkyl_copy_int_arr(pdim, idx, idxL);
    idxL[dir] = idx[dir] - 1;
    gkyl_copy_int_arr(pdim-cdim, idx_vel, idx_velL);
    idx_velL[0] = idx_velL[0]-1;
    long locL = gkyl_range_idx(&phase_range, idxL);
    long loc_velL = gkyl_range_idx(&up->vel_map->local_vel, idx_velL);
    const double *fL = (const double*) gkyl_array_cfetch(fin, locL);
    const double *fR = (const double*) gkyl_array_cfetch(fin, loc_phase);

    const double *vpL = (const double*) gkyl_array_cfetch(up->vel_map->vmap_prime, loc_velL);
    const double *vpR = (const double*) gkyl_array_cfetch(up->vel_map->vmap_prime, loc_vel);

    const struct gkyl_dg_vol_geom *dgv = gkyl_dg_geom_get_vol(up->dg_geom, idx);
    const struct gkyl_gk_dg_vol_geom *gkdgv = gkyl_gk_dg_geom_get_vol(up->gk_dg_geom, idx);

    cflrate_d[0] += up->flux_surfvpar[0](xc, up->phase_grid.dx, 
      vpL, vpR,
      vmap_d, vmapSq_d, up->charge, up->mass,
      dgv, gkdgv, bmag_d, phi_d,  fL, fR, flux_surf_d);
  }
}

// Host-side wrapper for gyrokinetic surface alpha calculation
void gkyl_dg_calc_gyrokinetic_vars_flux_surf_cu(struct gkyl_dg_calc_gyrokinetic_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range,
  const struct gkyl_range *conf_ext_range, const struct gkyl_range *phase_ext_range, const struct gkyl_array *phi, 
  const struct gkyl_array *fin, struct gkyl_array* flux_surf, struct gkyl_array* cflrate)
{
  gkyl_dg_calc_gyrokinetic_vars_flux_surf_cu_kernel<<<phase_range->volume, GKYL_DEFAULT_NUM_THREADS>>>(up->on_dev, 
    *conf_range, *phase_range, *conf_ext_range, *phase_ext_range, phi->on_dev, fin->on_dev,
    flux_surf->on_dev, cflrate->on_dev);

  struct gkyl_range vpar_range;
  int sublower[GKYL_MAX_DIM];
  int subupper[GKYL_MAX_DIM];
  for(int i = 0; i < up->pdim; i++) {
     sublower[i] = phase_range->lower[i];
     subupper[i] = phase_range->upper[i];
  }
  sublower[up->cdim] += 1;
  gkyl_sub_range_init(&vpar_range, phase_ext_range, sublower, subupper);
  gkyl_dg_calc_gyrokinetic_vars_flux_surfvpar_cu_kernel<<<vpar_range.volume, GKYL_DEFAULT_NUM_THREADS>>>(up->on_dev, 
    *conf_range, *phase_range, *conf_ext_range, *phase_ext_range, vpar_range, phi->on_dev, fin->on_dev,
    flux_surf->on_dev, cflrate->on_dev);

}

// CUDA kernel to set device pointers to gyrokinetic vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_gyrokinetic_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_gyrokinetic_vars *up, 
  int cdim, int vdim, int poly_order, enum gkyl_gkmodel_id gkmodel_id)
{
  if (gkmodel_id == GKYL_GK_MODEL_NO_BY) {
    for (int d=0; d<cdim; ++d) {
      up->flux_surf[d] = choose_gyrokinetic_flux_no_by_surf_conf_kern(d, cdim, vdim, poly_order);
      up->flux_edge_surf[d] = choose_gyrokinetic_flux_no_by_edge_surf_conf_kern(d, cdim, vdim, poly_order);
    }
    up->flux_surfvpar[0] = choose_gyrokinetic_flux_no_by_surf_vpar_kern(cdim, vdim, poly_order);
  }
  else {
    for (int d=0; d<cdim; ++d) {
      up->flux_surf[d] = choose_gyrokinetic_flux_surf_conf_kern(d, cdim, vdim, poly_order);
      up->flux_edge_surf[d] = choose_gyrokinetic_flux_edge_surf_conf_kern(d, cdim, vdim, poly_order);
    }
    up->flux_surfvpar[0] = choose_gyrokinetic_flux_surf_vpar_kern(cdim, vdim, poly_order);
  }
}

gkyl_dg_calc_gyrokinetic_vars*
gkyl_dg_calc_gyrokinetic_vars_cu_dev_new(const struct gkyl_rect_grid *phase_grid, 
  const struct gkyl_basis *conf_basis, const struct gkyl_basis *phase_basis, 
  const double charge, const double mass, enum gkyl_gkmodel_id gkmodel_id, 
  const struct gk_geometry *gk_geom, const struct gkyl_dg_geom *dg_geom, 
  const struct gkyl_gk_dg_geom *gk_dg_geom, const struct gkyl_velocity_map *vel_map)
{
  struct gkyl_dg_calc_gyrokinetic_vars *up = (struct gkyl_dg_calc_gyrokinetic_vars*) gkyl_malloc(sizeof(*up));

  up->phase_grid = *phase_grid;
  int cdim = conf_basis->ndim;
  int pdim = phase_basis->ndim;
  int vdim = pdim - cdim;
  int poly_order = phase_basis->poly_order;
  up->cdim = cdim;
  up->pdim = pdim;

  up->charge = charge;
  up->mass = mass;

  // Acquire pointers to on_dev objects so memcpy below copies those too.
  struct gk_geometry *geom_ho = gkyl_gk_geometry_acquire(gk_geom);
  struct gkyl_dg_geom *dg_geom_ho = gkyl_dg_geom_acquire(dg_geom);
  struct gkyl_gk_dg_geom *gk_dg_geom_ho = gkyl_gk_dg_geom_acquire(gk_dg_geom);
  struct gkyl_velocity_map *vel_map_ho = gkyl_velocity_map_acquire(vel_map);
  up->gk_geom = geom_ho->on_dev;
  up->dg_geom = dg_geom_ho->on_dev;
  up->gk_dg_geom = gk_dg_geom_ho->on_dev;
  up->vel_map = vel_map_ho->on_dev;

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_gyrokinetic_vars *up_cu = (struct gkyl_dg_calc_gyrokinetic_vars*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_gyrokinetic_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_gyrokinetic_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, cdim, vdim, poly_order, gkmodel_id);

  // set parent on_dev pointer
  up->on_dev = up_cu;

  // Updater should store host pointers.
  up->gk_geom = geom_ho; 
  up->dg_geom = dg_geom_ho; 
  up->gk_dg_geom = gk_dg_geom_ho; 
  up->vel_map = vel_map_ho; 
  
  return up;
}
