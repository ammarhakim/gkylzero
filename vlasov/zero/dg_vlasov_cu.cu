#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_vlasov.h>    
#include <gkyl_dg_vlasov_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_vlasov_set_auxfields_cu_kernel(const struct gkyl_dg_eqn *eqn, 
  const struct gkyl_array *field, const struct gkyl_array *cot_vec, 
  const struct gkyl_array *alpha_surf, const struct gkyl_array *sgn_alpha_surf, const struct gkyl_array *const_sgn_alpha)
{
  struct dg_vlasov *vlasov = container_of(eqn, struct dg_vlasov, eqn);
  vlasov->auxfields.field = field; 
  vlasov->auxfields.cot_vec = cot_vec; 
  vlasov->auxfields.alpha_surf = alpha_surf;
  vlasov->auxfields.sgn_alpha_surf = sgn_alpha_surf;
  vlasov->auxfields.const_sgn_alpha = const_sgn_alpha;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_vlasov_set_auxfields_cu(const struct gkyl_dg_eqn *eqn, struct gkyl_dg_vlasov_auxfields auxin)
{
  gkyl_vlasov_set_auxfields_cu_kernel<<<1,1>>>(eqn,
    auxin.field ? auxin.field->on_dev : 0,
    auxin.cot_vec ? auxin.cot_vec->on_dev : 0,
    auxin.alpha_surf ? auxin.alpha_surf->on_dev : 0,
    auxin.sgn_alpha_surf ? auxin.sgn_alpha_surf->on_dev : 0,
    auxin.const_sgn_alpha ? auxin.const_sgn_alpha->on_dev : 0);
}

// CUDA kernel to set device pointers to range object and vlasov kernel function
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_vlasov_set_cu_dev_ptrs(struct dg_vlasov *vlasov, enum gkyl_basis_type b_type,
  int cv_index, int cdim, int vdim, int poly_order, 
  enum gkyl_model_id model_id, enum gkyl_field_id field_id)
{
  vlasov->auxfields.field = 0;
  vlasov->auxfields.cot_vec = 0;
  vlasov->auxfields.alpha_surf = 0;
  vlasov->auxfields.sgn_alpha_surf = 0;
  vlasov->auxfields.const_sgn_alpha = 0;

  vlasov->eqn.surf_term = surf;
  vlasov->eqn.boundary_surf_term = boundary_surf;

  const gkyl_dg_vlasov_stream_vol_kern_list *stream_vol_kernels;
  const gkyl_dg_vlasov_vol_kern_list *vol_kernels;

  const gkyl_dg_vlasov_stream_surf_kern_list *stream_surf_x_kernels, 
    *stream_surf_y_kernels, 
    *stream_surf_z_kernels;

  const gkyl_dg_vlasov_accel_surf_kern_list *accel_surf_vx_kernels, 
    *accel_surf_vy_kernels, 
    *accel_surf_vz_kernels;

  const gkyl_dg_vlasov_stream_boundary_surf_kern_list *stream_boundary_surf_x_kernels, 
    *stream_boundary_surf_y_kernels,
    *stream_boundary_surf_z_kernels;
  
  const gkyl_dg_vlasov_accel_boundary_surf_kern_list *accel_boundary_surf_vx_kernels, 
    *accel_boundary_surf_vy_kernels,
    *accel_boundary_surf_vz_kernels;
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      stream_vol_kernels = ser_stream_vol_kernels;
      vol_kernels = ser_vol_kernels;

      stream_surf_x_kernels = ser_stream_surf_x_kernels;
      stream_surf_y_kernels = ser_stream_surf_y_kernels;
      stream_surf_z_kernels = ser_stream_surf_z_kernels;

      accel_surf_vx_kernels = ser_accel_surf_vx_kernels;
      accel_surf_vy_kernels = ser_accel_surf_vy_kernels;
      accel_surf_vz_kernels = ser_accel_surf_vz_kernels;

      stream_boundary_surf_x_kernels = ser_stream_boundary_surf_x_kernels;
      stream_boundary_surf_y_kernels = ser_stream_boundary_surf_y_kernels;
      stream_boundary_surf_z_kernels = ser_stream_boundary_surf_z_kernels;
      
      accel_boundary_surf_vx_kernels = ser_accel_boundary_surf_vx_kernels;
      accel_boundary_surf_vy_kernels = ser_accel_boundary_surf_vy_kernels;
      accel_boundary_surf_vz_kernels = ser_accel_boundary_surf_vz_kernels;
      
      break;

    case GKYL_BASIS_MODAL_TENSOR:
      stream_vol_kernels = tensor_stream_vol_kernels;
      vol_kernels = tensor_vol_kernels;

      stream_surf_x_kernels = tensor_stream_surf_x_kernels;
      stream_surf_y_kernels = tensor_stream_surf_y_kernels;
      stream_surf_z_kernels = tensor_stream_surf_z_kernels;

      accel_surf_vx_kernels = tensor_accel_surf_vx_kernels;
      accel_surf_vy_kernels = tensor_accel_surf_vy_kernels;
      accel_surf_vz_kernels = tensor_accel_surf_vz_kernels;

      stream_boundary_surf_x_kernels = tensor_stream_boundary_surf_x_kernels;
      stream_boundary_surf_y_kernels = tensor_stream_boundary_surf_y_kernels;
      stream_boundary_surf_z_kernels = tensor_stream_boundary_surf_z_kernels;

      accel_boundary_surf_vx_kernels = tensor_accel_boundary_surf_vx_kernels;
      accel_boundary_surf_vy_kernels = tensor_accel_boundary_surf_vy_kernels;
      accel_boundary_surf_vz_kernels = tensor_accel_boundary_surf_vz_kernels;
      
      break;  

    default:
      assert(false);
      break;    
  }
  if (field_id == GKYL_FIELD_NULL) {
    vlasov->eqn.vol_term = stream_vol_kernels[cv_index].kernels[poly_order];
  }
  else {
    vlasov->eqn.vol_term = vol_kernels[cv_index].kernels[poly_order];
    
    vlasov->accel_surf[0] = accel_surf_vx_kernels[cv_index].kernels[poly_order];
    if (vdim>1)
      vlasov->accel_surf[1] = accel_surf_vy_kernels[cv_index].kernels[poly_order];
    if (vdim>2)
      vlasov->accel_surf[2] = accel_surf_vz_kernels[cv_index].kernels[poly_order];
    
    vlasov->accel_boundary_surf[0] = accel_boundary_surf_vx_kernels[cv_index].kernels[poly_order];
    if (vdim>1)
      vlasov->accel_boundary_surf[1] = accel_boundary_surf_vy_kernels[cv_index].kernels[poly_order];
    if (vdim>2)
      vlasov->accel_boundary_surf[2] = accel_boundary_surf_vz_kernels[cv_index].kernels[poly_order];
  }
  // Streaming kernels are the same for each field_id
  vlasov->stream_surf[0] = stream_surf_x_kernels[cv_index].kernels[poly_order];
  if (cdim>1)
    vlasov->stream_surf[1] = stream_surf_y_kernels[cv_index].kernels[poly_order];
  if (cdim>2)
    vlasov->stream_surf[2] = stream_surf_z_kernels[cv_index].kernels[poly_order];
  
  vlasov->stream_boundary_surf[0] = stream_boundary_surf_x_kernels[cv_index].kernels[poly_order];
  if (cdim>1)
    vlasov->stream_boundary_surf[1] = stream_boundary_surf_y_kernels[cv_index].kernels[poly_order];
  if (cdim>2)
    vlasov->stream_boundary_surf[2] = stream_boundary_surf_z_kernels[cv_index].kernels[poly_order];   
}

struct gkyl_dg_eqn*
gkyl_dg_vlasov_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis,
  const struct gkyl_range* conf_range, const struct gkyl_range* phase_range,
  enum gkyl_model_id model_id, enum gkyl_field_id field_id)
{
  struct dg_vlasov *vlasov = (struct dg_vlasov*) gkyl_malloc(sizeof(struct dg_vlasov));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  vlasov->cdim = cdim;
  vlasov->pdim = pdim;

  vlasov->eqn.num_equations = 1;
  vlasov->conf_range = *conf_range;
  vlasov->phase_range = *phase_range;

  vlasov->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(vlasov->eqn.flags);
  vlasov->eqn.ref_count = gkyl_ref_count_init(gkyl_vlasov_free);

  // copy the host struct to device struct
  struct dg_vlasov *vlasov_cu = (struct dg_vlasov*) gkyl_cu_malloc(sizeof(struct dg_vlasov));
  gkyl_cu_memcpy(vlasov_cu, vlasov, sizeof(struct dg_vlasov), GKYL_CU_MEMCPY_H2D);

  dg_vlasov_set_cu_dev_ptrs<<<1,1>>>(vlasov_cu, cbasis->b_type, cv_index[cdim].vdim[vdim],
    cdim, vdim, poly_order, model_id, field_id);

  // set parent on_dev pointer
  vlasov->eqn.on_dev = &vlasov_cu->eqn;
  
  return &vlasov->eqn;
}
