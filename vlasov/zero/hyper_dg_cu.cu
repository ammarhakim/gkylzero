#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include "gkyl_alloc_flags_priv.h"
#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_array_ops.h>
#include <gkyl_hyper_dg.h>
#include <gkyl_hyper_dg_priv.h>
#include <gkyl_range.h>
#include <gkyl_rect_grid.h>
#include <gkyl_util.h>
}

__global__ static void
gkyl_hyper_dg_set_update_vol_cu_kernel(gkyl_hyper_dg *up, int update_vol_term)
{
  up->update_vol_term = update_vol_term;
}

__global__ static void
gkyl_hyper_dg_advance_cu_kernel(gkyl_hyper_dg* up, struct gkyl_range update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int ndim = up->ndim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  double xcl[GKYL_MAX_DIM], xcc[GKYL_MAX_DIM], xcr[GKYL_MAX_DIM];
  // integer used for selecting between left-edge zero-flux BCs and right-edge zero-flux BCs
  int edge;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < update_range.volume; linc1 += blockDim.x*gridDim.x) {
    // inverse index from linc1 to idxc
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idxc={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&update_range, linc1, idxc);
    gkyl_rect_grid_cell_center(&up->grid, idxc, xcc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&update_range, idxc);

    if (up->update_vol_term) {
      double cflr = up->equation->vol_term(
        up->equation, xcc, up->grid.dx, idxc,
        (const double*) gkyl_array_cfetch(fIn, linc), (double*) gkyl_array_fetch(rhs, linc)
      );
      double *cflrate_d = (double*) gkyl_array_fetch(cflrate, linc);
      cflrate_d[0] += cflr; // frequencies are additive
    }
    
    for (int d=0; d<up->num_up_dirs; ++d) {
      int dir = up->update_dirs[d];
      double cfls = 0.0;
      gkyl_copy_int_arr(ndim, idxc, idxl);
      gkyl_copy_int_arr(ndim, idxc, idxr);
      // TODO: fix for arbitrary subrange
      if ((up->zero_flux_flags[dir]      && idxc[dir] == update_range.lower[dir]) ||
          (up->zero_flux_flags[dir+ndim] && idxc[dir] == update_range.upper[dir])) {
        edge = (idxc[dir] == update_range.lower[dir]) ? -1 : 1;
        // use idxl to store interior edge index (first index away from skin cell)
        idxl[dir] = idxl[dir]-edge;

        gkyl_rect_grid_cell_center(&up->grid, idxl, xcl);
        long linl = gkyl_range_idx(&update_range, idxl);

        cfls = up->equation->boundary_surf_term(up->equation,
          dir, xcl, xcc, up->grid.dx, up->grid.dx,
          idxl, idxc, edge,
          (const double*) gkyl_array_cfetch(fIn, linl), (const double*) gkyl_array_cfetch(fIn, linc),
          (double*) gkyl_array_fetch(rhs, linc)
        );
      }
      else {
        idxl[dir] = idxl[dir]-1;
        idxr[dir] = idxr[dir]+1;
        gkyl_rect_grid_cell_center(&up->grid, idxl, xcl);
        gkyl_rect_grid_cell_center(&up->grid, idxr, xcr);
        long linl = gkyl_range_idx(&update_range, idxl); 
        long linr = gkyl_range_idx(&update_range, idxr);

        cfls = up->equation->surf_term(up->equation,
          dir, xcl, xcc, xcr, up->grid.dx, up->grid.dx, up->grid.dx,
          idxl, idxc, idxr,
          (const double*) gkyl_array_cfetch(fIn, linl), (const double*) gkyl_array_cfetch(fIn, linc),
          (const double*) gkyl_array_cfetch(fIn, linr), (double*) gkyl_array_fetch(rhs, linc)
        );
      }
      double *cflrate_d = (double*) gkyl_array_fetch(cflrate, linc);
      cflrate_d[0] += cfls; // frequencies are additive     
    }
  }
}

// wrapper to call advance kernel on device
void
gkyl_hyper_dg_advance_cu(gkyl_hyper_dg* up, const struct gkyl_range *update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int nblocks = update_range->nblocks;
  int nthreads = update_range->nthreads;

  gkyl_hyper_dg_advance_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *update_range,
    fIn->on_dev, cflrate->on_dev, rhs->on_dev);
}

void
gkyl_hyper_dg_set_update_vol_cu(gkyl_hyper_dg *up, int update_vol_term)
{
  gkyl_hyper_dg_set_update_vol_cu_kernel<<<1,1>>>(up, update_vol_term);
}

gkyl_hyper_dg*
gkyl_hyper_dg_cu_dev_new(const struct gkyl_rect_grid *grid,
  const struct gkyl_basis *basis, const struct gkyl_dg_eqn *equation,
  int num_up_dirs, int update_dirs[GKYL_MAX_DIM], int zero_flux_flags[2*GKYL_MAX_DIM],
  int update_vol_term)
{
  gkyl_hyper_dg *up = (gkyl_hyper_dg*) gkyl_malloc(sizeof(gkyl_hyper_dg));

  up->ndim = basis->ndim;
  up->num_basis = basis->num_basis;
  up->num_up_dirs = num_up_dirs;
  up->grid = *grid;

  for (int i=0; i<num_up_dirs; ++i)
    up->update_dirs[i] = update_dirs[i];

  for (int i=0; i<2*GKYL_MAX_DIM; ++i)
    up->zero_flux_flags[i] = zero_flux_flags[i];
    
  up->update_vol_term = update_vol_term;

  // aquire pointer to equation object
  struct gkyl_dg_eqn *eqn = gkyl_dg_eqn_acquire(equation);
  up->equation = eqn->on_dev; // this is so the memcpy below has eqn on_dev

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);
  
  // copy host struct to device struct
  gkyl_hyper_dg *up_cu = (gkyl_hyper_dg*) gkyl_cu_malloc(sizeof(gkyl_hyper_dg));
  gkyl_cu_memcpy(up_cu, up, sizeof(struct gkyl_hyper_dg), GKYL_CU_MEMCPY_H2D);
  up->on_dev = up_cu; // set parent pointer

  up->equation = eqn; // updater should store host pointer

  up->use_gpu = true;

  return up;
}
