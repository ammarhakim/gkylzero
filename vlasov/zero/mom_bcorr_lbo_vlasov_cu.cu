#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <assert.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_util.h>
#include <gkyl_mom_bcorr_lbo_vlasov.h>
#include <gkyl_mom_bcorr_lbo_vlasov_priv.h>
}

__global__
static void
gkyl_mom_bcorr_lbo_vlasov_set_cu_dev_ptrs(struct mom_type_bcorr_lbo_vlasov* mom_bcorr, enum gkyl_basis_type b_type, int vdim, int poly_order, int tblidx)
{
  mom_bcorr->momt.kernel = kernel;

  // choose kernel tables based on basis-function type
  const gkyl_mom_bcorr_lbo_vlasov_kern_list *mom_bcorr_lbo_vlasov_kernels;

  switch (b_type) {
  case GKYL_BASIS_MODAL_SERENDIPITY:
    mom_bcorr_lbo_vlasov_kernels = ser_mom_bcorr_lbo_vlasov_kernels;
    break;

  // case GKYL_BASIS_MODAL_TENSOR:
  //   mom_bcorr_lbo_vlasov_kernels = ten_mom_bcorr_lbo_vlasov_kernels;
  //   break;

  default:
    assert(false);
    break;
  }
  mom_bcorr->kernel = mom_bcorr_lbo_vlasov_kernels[tblidx].kernels[poly_order];
  mom_bcorr->momt.num_mom = vdim+1;
}

struct gkyl_mom_type*
gkyl_mom_bcorr_lbo_vlasov_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis, const double *vBoundary)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct mom_type_bcorr_lbo_vlasov *mom_bcorr = (struct mom_type_bcorr_lbo_vlasov*) gkyl_malloc(sizeof(struct mom_type_bcorr_lbo_vlasov));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  mom_bcorr->momt.cdim = cdim;
  mom_bcorr->momt.pdim = pdim;
  mom_bcorr->momt.poly_order = poly_order;
  mom_bcorr->momt.num_config = cbasis->num_basis;
  mom_bcorr->momt.num_phase = pbasis->num_basis;
  for (int d=0; d<vdim; ++d) {
    mom_bcorr->vBoundary[d] = vBoundary[d];
    mom_bcorr->vBoundary[d + vdim] = vBoundary[d + vdim];
  }
  mom_bcorr->momt.num_mom = vdim+1; // number of moments

  mom_bcorr->momt.flags = 0;
  GKYL_SET_CU_ALLOC(mom_bcorr->momt.flags);
  mom_bcorr->momt.ref_count = gkyl_ref_count_init(gkyl_mom_free);

  // copy struct to device
  struct mom_type_bcorr_lbo_vlasov *mom_bcorr_cu = (struct mom_type_bcorr_lbo_vlasov*)
    gkyl_cu_malloc(sizeof(struct mom_type_bcorr_lbo_vlasov));
  gkyl_cu_memcpy(mom_bcorr_cu, mom_bcorr, sizeof(struct mom_type_bcorr_lbo_vlasov), GKYL_CU_MEMCPY_H2D);

  assert(cv_index[cdim].vdim[vdim] != -1);


  gkyl_mom_bcorr_lbo_vlasov_set_cu_dev_ptrs<<<1,1>>>(mom_bcorr_cu, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  mom_bcorr->momt.on_dev = &mom_bcorr_cu->momt;

  return &mom_bcorr->momt;
}
