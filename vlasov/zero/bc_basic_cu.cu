#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_bc_basic.h>
#include <gkyl_bc_basic_priv.h>
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
}

__global__ static void
gkyl_bc_basic_create_set_cu_dev_ptrs(int dir, enum gkyl_edge_loc edge, int cdim, enum gkyl_bc_basic_type bctype,
  const struct gkyl_basis* basis, int ncomp, struct dg_bc_ctx *ctx, struct gkyl_array_copy_func *fout)
{
  ctx->dir = dir;
  ctx->edge = edge;
  ctx->cdim = cdim;
  ctx->basis = basis;
  ctx->ncomp = ncomp;

  switch (bctype) {
    case GKYL_BC_COPY:
    case GKYL_BC_FIXED_FUNC:
      fout->func = copy_bc;
      break;

    case GKYL_BC_ABSORB:
      fout->func = species_absorb_bc;
      break;

    case GKYL_BC_REFLECT:
      fout->func = reflect_bc;
      break;

    case GKYL_BC_DISTF_REFLECT:
      fout->func = species_reflect_bc;
      break;

    // Maxwell's perfect electrical conductor (zero normal B and zero tangent E)
    case GKYL_BC_MAXWELL_PEC:
      fout->func = maxwell_pec_bc;
      break;

    // Maxwell's symmetry BC (zero normal E and zero tangent B)
    case GKYL_BC_MAXWELL_SYM:
      fout->func = maxwell_sym_bc;
      break;

    // Reservoir Maxwell's BCs for heat flux problem
    // Based on Roberg-Clark et al. PRL 2018
    // NOTE: ONLY WORKS WITH X BOUNDARY 
    case GKYL_BC_MAXWELL_RESERVOIR:
      fout->func = maxwell_reservoir_bc;
      break;

    // PKPM Reflecting wall for distribution function
    case GKYL_BC_PKPM_SPECIES_REFLECT:
      fout->func = pkpm_species_reflect_bc;
      break;    

    // PKPM Reflecting wall for momentum
    case GKYL_BC_PKPM_MOM_REFLECT:
      fout->func = pkpm_mom_reflect_bc;
      break;    

    // PKPM No-slip wall for momentum
    case GKYL_BC_PKPM_MOM_NO_SLIP:
      fout->func = pkpm_mom_no_slip_bc;
      break; 

    // Euler Reflecting wall 
    case GKYL_BC_EULER_REFLECT:
      fout->func = euler_reflect_bc;
      break;    

    // Euler No-slip wall 
    case GKYL_BC_EULER_NO_SLIP:
      fout->func = euler_no_slip_bc;
      break;  


    case GKYL_BC_CONF_BOUNDARY_VALUE:
      fout->func = conf_boundary_value_bc;
      break;

    default:
      assert(false);
      break;
  }
  fout->ctx = ctx;
}

struct gkyl_array_copy_func*
gkyl_bc_basic_create_arr_copy_func_cu(int dir, enum gkyl_edge_loc edge, int cdim, enum gkyl_bc_basic_type bctype,
  const struct gkyl_basis *basis, int ncomp)
{
  // create host context and bc func structs
  struct dg_bc_ctx *ctx = (struct dg_bc_ctx*) gkyl_malloc(sizeof(struct dg_bc_ctx));
  struct gkyl_array_copy_func *fout = (struct gkyl_array_copy_func*) gkyl_malloc(sizeof(struct gkyl_array_copy_func));
  fout->ctx = ctx;

  fout->flags = 0;
  GKYL_SET_CU_ALLOC(fout->flags);

  // create device context and bc func structs
  struct dg_bc_ctx *ctx_cu = (struct dg_bc_ctx*) gkyl_cu_malloc(sizeof(struct dg_bc_ctx));
  struct gkyl_array_copy_func *fout_cu = (struct gkyl_array_copy_func*) gkyl_cu_malloc(sizeof(struct gkyl_array_copy_func));

  gkyl_cu_memcpy(ctx_cu, ctx, sizeof(struct dg_bc_ctx), GKYL_CU_MEMCPY_H2D);
  gkyl_cu_memcpy(fout_cu, fout, sizeof(struct gkyl_array_copy_func), GKYL_CU_MEMCPY_H2D);

  fout->ctx_on_dev = ctx_cu;

  gkyl_bc_basic_create_set_cu_dev_ptrs<<<1,1>>>(dir, edge, cdim, bctype, basis, ncomp, ctx_cu, fout_cu);

  // set parent on_dev pointer
  fout->on_dev = fout_cu;
  return fout;
}
