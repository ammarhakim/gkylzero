#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_diffusion_vlasov.h>    
#include <gkyl_dg_diffusion_vlasov_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_dg_diffusion_vlasov_set_auxfields_cu_kernel(const struct gkyl_dg_eqn* eqn, const struct gkyl_array* D)
{
  struct dg_diffusion_vlasov* diffusion = container_of(eqn, struct dg_diffusion_vlasov, eqn);
  diffusion->auxfields.D = D;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_dg_diffusion_vlasov_set_auxfields_cu(const struct gkyl_dg_eqn* eqn, struct gkyl_dg_diffusion_vlasov_auxfields auxin)
{
  gkyl_dg_diffusion_vlasov_set_auxfields_cu_kernel<<<1,1>>>(eqn, auxin.D->on_dev);
}

__global__ void static
dg_diffusion_vlasov_set_cu_dev_ptrs(struct dg_diffusion_vlasov *diffusion, enum gkyl_basis_type b_type, int cdim, int vdim, int poly_order, int diff_order, int diffdirs_linidx)
{
  diffusion->auxfields.D = 0; 

  const gkyl_dg_diffusion_vlasov_vol_kern_list *vol_kernels;
  const gkyl_dg_diffusion_vlasov_surf_kern_list *surfx_kernels;
  const gkyl_dg_diffusion_vlasov_surf_kern_list *surfy_kernels;
  const gkyl_dg_diffusion_vlasov_surf_kern_list *surfz_kernels;
  const gkyl_dg_diffusion_vlasov_boundary_surf_kern_list *boundary_surfx_kernels;
  const gkyl_dg_diffusion_vlasov_boundary_surf_kern_list *boundary_surfy_kernels;
  const gkyl_dg_diffusion_vlasov_boundary_surf_kern_list *boundary_surfz_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels            = diffusion->const_coeff? ser_vol_kernels_constcoeff                   : ser_vol_kernels_varcoeff                  ;
      surfx_kernels          = diffusion->const_coeff? ser_vlasov_surfx_kernels_constcoeff          : ser_vlasov_surfx_kernels_varcoeff         ;
      surfy_kernels          = diffusion->const_coeff? ser_vlasov_surfy_kernels_constcoeff          : ser_vlasov_surfy_kernels_varcoeff         ;
      surfz_kernels          = diffusion->const_coeff? ser_vlasov_surfz_kernels_constcoeff          : ser_vlasov_surfz_kernels_varcoeff         ;
      boundary_surfx_kernels = diffusion->const_coeff? ser_vlasov_boundary_surfx_kernels_constcoeff : ser_vlasov_boundary_surfx_kernels_varcoeff;
      boundary_surfy_kernels = diffusion->const_coeff? ser_vlasov_boundary_surfy_kernels_constcoeff : ser_vlasov_boundary_surfy_kernels_varcoeff;
      boundary_surfz_kernels = diffusion->const_coeff? ser_vlasov_boundary_surfz_kernels_constcoeff : ser_vlasov_boundary_surfz_kernels_varcoeff;
      break;

    default:
      assert(false);
      break;
  }

  diffusion->eqn.num_equations = 1;
  diffusion->eqn.surf_term = surf;
  diffusion->eqn.boundary_surf_term = boundary_surf;

  diffusion->eqn.vol_term = CKVOL(vol_kernels, cdim, diff_order, poly_order, diffdirs_linidx);

  diffusion->surf[0] = CKSURF(surfx_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>1)
    diffusion->surf[1] = CKSURF(surfy_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>2)
    diffusion->surf[2] = CKSURF(surfz_kernels, diff_order, cdim, vdim, poly_order);

  diffusion->boundary_surf[0] = CKSURF(boundary_surfx_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>1)
    diffusion->boundary_surf[1] = CKSURF(boundary_surfy_kernels, diff_order, cdim, vdim, poly_order);
  if (cdim>2)
    diffusion->boundary_surf[2] = CKSURF(boundary_surfz_kernels, diff_order, cdim, vdim, poly_order);
}

struct gkyl_dg_eqn*
gkyl_dg_diffusion_vlasov_cu_dev_new(const struct gkyl_basis *basis, const struct gkyl_basis *cbasis,
  bool is_diff_const, const bool *diff_in_dir, int diff_order, const struct gkyl_range *diff_range)
{
  struct dg_diffusion_vlasov* diffusion = (struct dg_diffusion_vlasov*) gkyl_malloc(sizeof(struct dg_diffusion_vlasov));

  int cdim = cbasis->ndim;
  int vdim = basis->ndim - cdim;
  int poly_order = cbasis->poly_order;

  diffusion->const_coeff = is_diff_const;
  diffusion->num_basis = basis->num_basis;
  for (int d=0; d<cdim; d++) diffusion->diff_in_dir[d] = diff_in_dir[d];

  int dirs_linidx = diffdirs_linidx(diff_in_dir, cdim);

  diffusion->diff_range = *diff_range;

  diffusion->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(diffusion->eqn.flags);
  diffusion->eqn.ref_count = gkyl_ref_count_init(gkyl_dg_diffusion_vlasov_free);

  // copy the host struct to device struct
  struct dg_diffusion_vlasov* diffusion_cu = (struct dg_diffusion_vlasov*) gkyl_cu_malloc(sizeof(struct dg_diffusion_vlasov));
  gkyl_cu_memcpy(diffusion_cu, diffusion, sizeof(struct dg_diffusion_vlasov), GKYL_CU_MEMCPY_H2D);
  dg_diffusion_vlasov_set_cu_dev_ptrs<<<1,1>>>(diffusion_cu, cbasis->b_type, cdim, vdim, poly_order, diff_order, dirs_linidx);

  // set parent on_dev pointer
  diffusion->eqn.on_dev = &diffusion_cu->eqn;

  return &diffusion->eqn;
}
