#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_positivity_shift_vlasov.h>
#include <gkyl_positivity_shift_vlasov_priv.h>
#include <gkyl_array_ops.h>
#include <float.h>
}

// CUDA kernel to set device pointers to kernels.
__global__ static void
gkyl_pos_shift_vlasov_set_cu_ker_ptrs(struct gkyl_positivity_shift_vlasov_kernels *kernels,
  struct gkyl_basis cbasis, struct gkyl_basis pbasis, enum gkyl_positivity_shift_type stype)
{
  int cdim = cbasis.ndim, pdim = pbasis.ndim;
  int vdim = pdim-cdim;
  enum gkyl_basis_type cbasis_type = cbasis.b_type, pbasis_type = pbasis.b_type;
  int poly_order = pbasis.poly_order;

  int plin = pos_shift_vlasov_cv_index[cdim].vdim[vdim];

  switch (pbasis_type) {
    case GKYL_BASIS_MODAL_TENSOR:
      kernels->is_m0_positive = pos_shift_vlasov_kern_list_m0_pos_check_tensor[cdim-1].kernels[poly_order-1];
      kernels->shift = stype == GKYL_POSITIVITY_SHIFT_TYPE_SHIFT_ONLY?
        pos_shift_vlasov_kern_list_shift_tensor[plin].kernels[poly_order-1] :
        pos_shift_vlasov_kern_list_MRSlimiter_tensor[plin].kernels[poly_order-1];
      kernels->m0 = pos_shift_vlasov_kern_list_m0_tensor[plin].kernels[poly_order-1];
      kernels->conf_phase_mul_op = choose_mul_conf_phase_kern(pbasis_type, cdim, vdim, poly_order);
      break;
    default:
      assert(false);
      break;
  }

  switch (cbasis_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      kernels->conf_inv_op = choose_ser_inv_kern(cdim, poly_order);
      kernels->conf_mul_op = choose_ser_mul_kern(cdim, poly_order);
      break;
    default:
      assert(false);
      break;
  }
};

void
pos_shift_vlasov_choose_shift_kernel_cu(struct gkyl_positivity_shift_vlasov_kernels *kernels,
  struct gkyl_basis cbasis, struct gkyl_basis pbasis, enum gkyl_positivity_shift_type stype)
{
  gkyl_pos_shift_vlasov_set_cu_ker_ptrs<<<1,1>>>(kernels, cbasis, pbasis, stype);
}

// Function borrowed from array_reduce_cu.cu.
__device__ static __forceinline__ double
pos_shift_atomicMax_double(double *address, double val)
{
  unsigned long long int ret = __double_as_longlong(*address);
  while(val > __longlong_as_double(ret))
  {
    unsigned long long int old = ret;
    if((ret = atomicCAS((unsigned long long int*)address, old, __double_as_longlong(val))) == old)
      break;
  }
  return __longlong_as_double(ret);
}

__global__ void
gkyl_positivity_shift_vlasov_advance_int_array_clear_cu_ker(struct gkyl_array* out, int val)
{
  int *out_d = (int*) out->data;
  unsigned long start_id = threadIdx.x + blockIdx.x*blockDim.x;
  unsigned long nelm = out->size*out->ncomp;
  for (unsigned long linc = start_id; linc < nelm; linc += blockDim.x*gridDim.x)
    out_d[linc] = val;
}

__global__ static void
gkyl_positivity_shift_vlasov_advance_shift_cu_ker(
  struct gkyl_positivity_shift_vlasov_kernels *kers, const struct gkyl_rect_grid grid,
  const struct gkyl_range conf_range, const struct gkyl_range phase_range,
  double *ffloor, double ffloor_fac, double cellav_fac, struct gkyl_array* GKYL_RESTRICT shiftedf,
  struct gkyl_array* GKYL_RESTRICT distf, struct gkyl_array* GKYL_RESTRICT m0, struct gkyl_array* GKYL_RESTRICT delta_m0)
{
  int pidx[GKYL_MAX_DIM];
   double xc[GKYL_MAX_DIM];

  double distf_max = -DBL_MAX;
  int cdim = conf_range.ndim, pdim = phase_range.ndim;

  const int num_cbasis = 20; // MF 2024/09/03: Hardcoded to p=2 3x ser for now.

  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < phase_range.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&phase_range, tid, pidx);

    long clinidx = gkyl_range_idx(&conf_range, pidx);
    long plinidx = gkyl_range_idx(&phase_range, pidx);

    gkyl_rect_grid_cell_center(&grid, pidx, xc);

    int *shiftedf_c = (int*) gkyl_array_fetch(shiftedf, clinidx);
    double *m0_c = (double*) gkyl_array_fetch(m0, clinidx);
    double *delta_m0_c = (double*) gkyl_array_fetch(delta_m0, clinidx);
    double *distf_c = (double*) gkyl_array_fetch(distf, plinidx);

    // Contribution to the old number density from this v-space cell.
    double m0Local_in[num_cbasis];
    for (unsigned int k=0; k<delta_m0->ncomp; ++k)
      m0Local_in[k] = 0.0;
    kers->m0(xc, grid.dx, pidx, distf_c, m0Local_in);

    // Add to the old number density.
    for (unsigned int k = 0; k < delta_m0->ncomp; ++k)
      atomicAdd(&delta_m0_c[k], m0Local_in[k]);

    // Shift f if needed.
    bool shifted_node = kers->shift(ffloor[0], distf_c);

    if (shifted_node) {
      // Compute the new number density local to this phase-space cell.
      double m0Local_out[num_cbasis];
      for (unsigned int k=0; k<m0->ncomp; ++k)
        m0Local_out[k] = 0.0;
      kers->m0(xc, grid.dx, pidx, distf_c, m0Local_out);

      if (kers->is_m0_positive(m0Local_in)) {
        // Rescale f in this cell so it keeps the same density.
        double m0ratio_c[num_cbasis];
        kers->conf_inv_op(m0Local_out, m0ratio_c);
        kers->conf_mul_op(m0Local_in, m0ratio_c, m0ratio_c);

        kers->conf_phase_mul_op(m0ratio_c, distf_c, distf_c);

        // Add contribution from this phase-space cell to the new number density.
        for (unsigned int k = 0; k < m0->ncomp; ++k)
          atomicAdd(&m0_c[k], m0Local_in[k]);
      }
      else {
        // Add contribution from this phase-space cell to the new number density.
        for (unsigned int k = 0; k < m0->ncomp; ++k)
          atomicAdd(&m0_c[k], m0Local_out[k]);

        atomicOr(shiftedf_c, shifted_node);
      }
    }
    else {
      // Add contribution from this phase-space cell to the new number density.
      for (unsigned int k = 0; k < m0->ncomp; ++k)
        atomicAdd(&m0_c[k], m0Local_in[k]);
    }

    distf_max = fmax(distf_max, distf_c[0]);

  }

  pos_shift_atomicMax_double(ffloor, ffloor_fac * distf_max * cellav_fac);
}

__global__ static void
gkyl_positivity_shift_vlasov_advance_scalef_cu_ker(
  struct gkyl_positivity_shift_vlasov_kernels *kers,
  const struct gkyl_range conf_range, const struct gkyl_range phase_range,
  const struct gkyl_array* GKYL_RESTRICT shiftedf, const struct gkyl_array* GKYL_RESTRICT m0,
  const struct gkyl_array* GKYL_RESTRICT delta_m0, struct gkyl_array* GKYL_RESTRICT distf)
{
  int pidx[GKYL_MAX_DIM];

  const int num_cbasis = 20; // MF 2024/09/03: Hardcoded to p=2 3x ser for now.

  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < phase_range.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&phase_range, tid, pidx);

    long clinidx = gkyl_range_idx(&conf_range, pidx);
    const int *shiftedf_c = (const int*) gkyl_array_cfetch(shiftedf, clinidx);

    if (shiftedf_c[0]) {
      const double *delta_m0_c = (const double*) gkyl_array_cfetch(delta_m0, clinidx);
      if (kers->is_m0_positive(delta_m0_c)) {
        // Rescale f so it has the same m0 at this conf-space cell.
        const double *m0_c = (const double*) gkyl_array_cfetch(m0, clinidx);
        double m0ratio_c[num_cbasis];
        kers->conf_inv_op(m0_c, m0ratio_c);
        kers->conf_mul_op(delta_m0_c, m0ratio_c, m0ratio_c);

        long plinidx = gkyl_range_idx(&phase_range, pidx);
        double *distf_c = (double*) gkyl_array_fetch(distf, plinidx);
        kers->conf_phase_mul_op(m0ratio_c, distf_c, distf_c);
      }
    }
  }
}

__global__ static void
gkyl_positivity_shift_vlasov_advance_m0fix_cu_ker(
  struct gkyl_positivity_shift_vlasov_kernels *kers,
  const struct gkyl_range conf_range, const struct gkyl_array* GKYL_RESTRICT shiftedf,
  struct gkyl_array* GKYL_RESTRICT m0, struct gkyl_array* GKYL_RESTRICT delta_m0)
{
  int cidx[GKYL_MAX_CDIM];

  for(unsigned long tid = threadIdx.x + blockIdx.x*blockDim.x;
      tid < conf_range.volume; tid += blockDim.x*gridDim.x) {
    gkyl_sub_range_inv_idx(&conf_range, tid, cidx);

    long clinidx = gkyl_range_idx(&conf_range, cidx);

    const int *shiftedf_c = (const int*) gkyl_array_cfetch(shiftedf, clinidx);
    double *delta_m0_c = (double*) gkyl_array_fetch(delta_m0, clinidx);

    if (shiftedf_c[0]) {
      double *m0_c = (double*) gkyl_array_fetch(m0, clinidx);
      if (kers->is_m0_positive(delta_m0_c)) {
	for (int k=0; k<m0->ncomp; k++) {
          m0_c[k] = delta_m0_c[k];
          delta_m0_c[k] = 0.0;
	}
      }
      else {
	for (int k=0; k<m0->ncomp; k++)
          delta_m0_c[k] = m0_c[k] - delta_m0_c[k];
      }
    }
    else {
      for (int k=0; k<m0->ncomp; k++)
        delta_m0_c[k] = 0.0;
    }
  }
}

void
gkyl_positivity_shift_vlasov_advance_cu(gkyl_positivity_shift_vlasov* up,
  const struct gkyl_range *conf_rng, const struct gkyl_range *phase_rng,
  struct gkyl_array *GKYL_RESTRICT distf, struct gkyl_array *GKYL_RESTRICT m0,
  struct gkyl_array *GKYL_RESTRICT delta_m0)
{
  int nblocks_phase = phase_rng->nblocks, nthreads_phase = phase_rng->nthreads;
  int nblocks_conf = conf_rng->nblocks, nthreads_conf = conf_rng->nthreads;

  gkyl_array_clear_range(m0, 0.0, conf_rng);
  gkyl_array_clear_range(delta_m0, 0.0, conf_rng);

  // Set shiftedf boolean (int) to 0s.
  gkyl_positivity_shift_vlasov_advance_int_array_clear_cu_ker<<<nblocks_conf, nthreads_conf>>>
    (up->shiftedf->on_dev, 0);

  // Shift f is needed & scale f locally if initial local contribution to M0 was >0.
  gkyl_positivity_shift_vlasov_advance_shift_cu_ker<<<nblocks_phase, nthreads_phase>>>
    (up->kernels, up->grid, *conf_rng, *phase_rng, up->ffloor, up->ffloor_fac,
     up->cellav_fac, up->shiftedf->on_dev, distf->on_dev, m0->on_dev, delta_m0->on_dev);

  // If a shift took place, rescale f so it keeps the same M0.
  gkyl_positivity_shift_vlasov_advance_scalef_cu_ker<<<nblocks_phase, nthreads_phase>>>
    (up->kernels, *conf_rng, *phase_rng, up->shiftedf->on_dev, m0->on_dev, delta_m0->on_dev, distf->on_dev);

  // Ensure m0 and delta_m0 are correct based on whether a shift took place.
  gkyl_positivity_shift_vlasov_advance_m0fix_cu_ker<<<nblocks_conf, nthreads_conf>>>
    (up->kernels, *conf_rng, up->shiftedf->on_dev, m0->on_dev, delta_m0->on_dev);
}
