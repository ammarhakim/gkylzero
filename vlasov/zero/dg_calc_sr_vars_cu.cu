#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_calc_sr_vars.h>
#include <gkyl_dg_calc_sr_vars_priv.h>
#include <gkyl_util.h>
}

__global__ void
gkyl_calc_sr_vars_init_p_vars_cu_kernel(gkyl_dg_calc_sr_vars* up, 
  struct gkyl_array* gamma, struct gkyl_array* gamma_inv)
{
  int idx[GKYL_MAX_DIM];
  // Cell center array
  double xc[GKYL_MAX_DIM];  

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < up->vel_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&up->vel_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->vel_grid, idx, xc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&up->vel_range, idx);

    double *gamma_d = (double*) gkyl_array_fetch(gamma, loc);
    double *gamma_inv_d = (double*) gkyl_array_fetch(gamma_inv, loc);
    up->sr_p_vars(xc, up->vel_grid.dx, gamma_d, gamma_inv_d);
  }
}

// Host-side wrapper for initialization of momentum variables (gamma, gamma_inv) 
void
gkyl_calc_sr_vars_init_p_vars_cu(struct gkyl_dg_calc_sr_vars *up, 
  struct gkyl_array* gamma, struct gkyl_array* gamma_inv)
{
  int nblocks = up->vel_range.nblocks;
  int nthreads = up->vel_range.nthreads;
  gkyl_calc_sr_vars_init_p_vars_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    gamma->on_dev, gamma_inv->on_dev);
}

__global__ static void
gkyl_dg_calc_sr_vars_n_set_cu_kernel(gkyl_dg_calc_sr_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* M0, const struct gkyl_array* M1i)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    const double *M0_d = (const double*) gkyl_array_cfetch(M0, loc);
    const double *M1i_d = (const double*) gkyl_array_cfetch(M1i, loc);

    up->sr_n_set(count, As, xs, M0_d, M1i_d);
  }
}

__global__ static void
gkyl_dg_calc_sr_vars_n_copy_cu_kernel(gkyl_dg_calc_sr_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* M0, struct gkyl_array* n)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    const double *M0_d = (const double*) gkyl_array_cfetch(M0, loc);
    double* n_d = (double*) gkyl_array_fetch(n, loc);

    up->sr_n_copy(count, xs, M0_d, n_d);
  }
}

// Host-side wrapper for SR rest-frame density calculation
void gkyl_dg_calc_sr_vars_n_cu(struct gkyl_dg_calc_sr_vars *up, 
  const struct gkyl_array* M0, const struct gkyl_array* M1i, struct gkyl_array* n)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_sr_vars_n_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As->on_dev, up->xs->on_dev, conf_range,
    M0->on_dev, M1i->on_dev);

  if (up->poly_order > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem, up->As, up->xs);
    assert(status);
  }

  gkyl_dg_calc_sr_vars_n_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs->on_dev, conf_range, M0->on_dev, n->on_dev);
}

__global__ void
gkyl_dg_calc_sr_vars_GammaV_cu_kernel(struct gkyl_dg_calc_sr_vars *up, 
  struct gkyl_range conf_range,
  const struct gkyl_array* u_i, struct gkyl_array* u_i_sq, 
  struct gkyl_array* GammaV, struct gkyl_array* GammaV_sq)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);

    const double *u_i_d = (const double*) gkyl_array_cfetch(u_i, loc_conf);
    double *u_i_sq_d = (double*) gkyl_array_fetch(u_i_sq, loc_conf);
    double *GammaV_d = (double*) gkyl_array_fetch(GammaV, loc_conf);
    double *GammaV_sq_d = (double*) gkyl_array_fetch(GammaV_sq, loc_conf);

    up->sr_GammaV(u_i_d, u_i_sq_d, GammaV_d, GammaV_sq_d); 
  }
}

// Host-side wrapper for derived quantities from bulk four-velocity
void
gkyl_dg_calc_sr_vars_GammaV_cu(struct gkyl_dg_calc_sr_vars *up, 
  const struct gkyl_range *conf_range,
  const struct gkyl_array* u_i, struct gkyl_array* u_i_sq, 
  struct gkyl_array* GammaV, struct gkyl_array* GammaV_sq)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_sr_vars_GammaV_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    u_i->on_dev, u_i_sq->on_dev, GammaV->on_dev, GammaV_sq->on_dev);
}

__global__ void
gkyl_dg_calc_sr_vars_pressure_cu_kernel(struct gkyl_dg_calc_sr_vars *up, 
  struct gkyl_range conf_range, struct gkyl_range phase_range, 
  const struct gkyl_array* gamma, const struct gkyl_array* gamma_inv, 
  const struct gkyl_array* u_i, const struct gkyl_array* u_i_sq, 
  const struct gkyl_array* GammaV, const struct gkyl_array* GammaV_sq, 
  const struct gkyl_array* f, struct gkyl_array* sr_pressure)
{
  int cdim = conf_range.ndim;
  int pdim = phase_range.ndim;

  double xc[GKYL_MAX_DIM];
  int idx_vel[GKYL_MAX_DIM];
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < phase_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&phase_range, linc1, idx);
    gkyl_rect_grid_cell_center(&up->phase_grid, idx, xc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc_conf = gkyl_range_idx(&conf_range, idx);
    long loc_phase = gkyl_range_idx(&phase_range, idx);

    for (int i=0; i<pdim-cdim; ++i) {
      idx_vel[i] = idx[cdim+i];
    }
    long loc_vel = gkyl_range_idx(&up->vel_range, idx_vel);

    const double *gamma_d = (const double*) gkyl_array_cfetch(gamma, loc_vel);
    const double *gamma_inv_d = (const double*) gkyl_array_cfetch(gamma_inv, loc_vel);
    const double *u_i_d = (const double*) gkyl_array_cfetch(u_i, loc_conf);
    const double *u_i_sq_d = (const double*) gkyl_array_cfetch(u_i_sq, loc_conf);
    const double *GammaV_d = (const double*) gkyl_array_cfetch(GammaV, loc_conf);
    const double *GammaV_sq_d = (const double*) gkyl_array_cfetch(GammaV_sq, loc_conf);
    const double *f_d = (const double*) gkyl_array_cfetch(f, loc_phase);

    double momLocal[32]; // hard-coded to max confBasis.num_basis (3x p=3 Ser) for now.
    for (unsigned int k=0; k<32; ++k) {
      momLocal[k] = 0.0;
    }

    up->sr_pressure(xc, up->phase_grid.dx, 
      gamma_d, gamma_inv_d, u_i_d, u_i_sq_d, GammaV_d, GammaV_sq_d, 
      f_d, &momLocal[0]);  

    // Accumulate output to output array atomically to avoid race conditions
    double *sr_pressure_d = (double*) gkyl_array_fetch(sr_pressure, loc_conf);
    for (unsigned int k = 0; k < sr_pressure->ncomp; ++k) {
      atomicAdd(&sr_pressure_d[k], momLocal[k]);
    }   
  }  
}
// Host-side wrapper for rest-frame pressure calculation
void 
gkyl_dg_calc_sr_vars_pressure_cu(struct gkyl_dg_calc_sr_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_range *phase_range, 
  const struct gkyl_array* gamma, const struct gkyl_array* gamma_inv, 
  const struct gkyl_array* u_i, const struct gkyl_array* u_i_sq, 
  const struct gkyl_array* GammaV, const struct gkyl_array* GammaV_sq, 
  const struct gkyl_array* f, struct gkyl_array* sr_pressure)
{
  int nblocks = phase_range->nblocks;
  int nthreads = phase_range->nthreads;
  gkyl_array_clear(sr_pressure, 0.0); 
  gkyl_dg_calc_sr_vars_pressure_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, 
    *conf_range, *phase_range, 
    gamma->on_dev, gamma_inv->on_dev, 
    u_i->on_dev, u_i_sq->on_dev, GammaV->on_dev, GammaV_sq->on_dev, 
    f->on_dev, sr_pressure->on_dev);
}

// CUDA kernel to set device pointers to sr vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_sr_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_sr_vars *up, 
  enum gkyl_basis_type b_type, enum gkyl_basis_type b_type_v,
  int cdim, int vdim, int poly_order, int poly_order_v)
{
  up->sr_p_vars = choose_sr_p_vars_kern(b_type_v, vdim, poly_order_v);
  up->sr_n_set = choose_sr_vars_n_set_kern(b_type, cdim, vdim, poly_order);
  up->sr_n_copy = choose_sr_vars_n_copy_kern(b_type, cdim, vdim, poly_order);
  up->sr_GammaV = choose_sr_vars_GammaV_kern(b_type, cdim, vdim, poly_order);
  up->sr_pressure = choose_sr_vars_pressure_kern(b_type, cdim, vdim, poly_order);
}

gkyl_dg_calc_sr_vars*
gkyl_dg_calc_sr_vars_cu_dev_new(const struct gkyl_rect_grid *phase_grid, const struct gkyl_rect_grid *vel_grid, 
  const struct gkyl_basis *conf_basis, const struct gkyl_basis *vel_basis, 
  const struct gkyl_range *mem_range, const struct gkyl_range *vel_range)
{
  struct gkyl_dg_calc_sr_vars *up = (struct gkyl_dg_calc_sr_vars*) gkyl_malloc(sizeof(*up));

  up->phase_grid = *phase_grid;
  up->vel_grid = *vel_grid;
  up->vel_range = *vel_range;

  int nc = conf_basis->num_basis;
  int cdim = conf_basis->ndim;
  int poly_order = conf_basis->poly_order;
  enum gkyl_basis_type b_type = conf_basis->b_type;
  // store polynomial order and mem_range for linear solve
  up->poly_order = poly_order;
  up->mem_range = *mem_range;

  int vdim = vel_basis->ndim;
  int poly_order_v = vel_basis->poly_order;
  enum gkyl_basis_type b_type_v = vel_basis->b_type;

  // Linear system for solving for the drift velocity V_drift = M1i/M0 
  // and then computing the rest-frame density n = GammaV_inv*M0 
  // where GammaV_inv = sqrt(1 - |V_drift|^2)
  up->Ncomp = vdim; 
  up->As = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, nc);
  up->xs = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, 1);
  up->mem = gkyl_nmat_linsolve_lu_cu_dev_new(up->As->num, up->As->nr);

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_sr_vars *up_cu = (struct gkyl_dg_calc_sr_vars*) gkyl_cu_malloc(sizeof(*up_cu));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_sr_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_sr_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, b_type, b_type_v, cdim, vdim, poly_order, poly_order_v);

  // set parent on_dev pointer
  up->on_dev = up_cu;
  
  return up;
}
